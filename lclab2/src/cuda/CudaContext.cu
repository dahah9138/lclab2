#include "CudaContext.cuh"
#include "CudaContext.h"

namespace LC { namespace Cuda {

    // Utilities

    int DeviceAllocate(void** data, unsigned int size) {
        return checkCuda(hipMalloc(data, size));
    }

    int DeviceAllocateManaged(void** data, unsigned int size) {
        return checkCuda(hipMallocManaged(data, size));
    }

    int DeviceAllocatePinned(void** data, unsigned int size) {
        return checkCuda(hipHostAlloc(data, size, 0));
    }

    int Free(void* data) {
        return checkCuda(hipFree(data));
    }

    int Sync() {
        return checkCuda(hipDeviceSynchronize());
    }

    int Memcpy(void* dst, void* src, unsigned int size, Transfer kind) {
        return checkCuda(hipMemcpy(dst, src, size, static_cast<hipMemcpyKind>(static_cast<int>(kind))));
    }

    int CopyFromSymbol(void *dst, const void *sym, unsigned int size, Transfer kind) {
        return checkCuda(hipMemcpyFromSymbol(dst, HIP_SYMBOL(sym), size, static_cast<hipMemcpyKind>(static_cast<int>(kind))));
    }

    int CopyToSymbol(const void* dst, const void* sym, unsigned int size, Transfer kind) {
        return checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(dst), sym, size, static_cast<hipMemcpyKind>(static_cast<int>(kind))));
    }




}}