#include "hip/hip_runtime.h"
#include "CudaContext.cuh"
#include "scalar.h"

namespace LC { namespace FrankOseen { namespace ElasticOnly { namespace RBF {

	// Tuple of derivatives

	struct Director {
		scalar nx = 0.0, ny = 0.0, nz = 0.0;
	};

	struct OneConstDerivatives {
		Director Dx, Dy, Dz, Dlap;
	};

	HEMI_DEV_CALLABLE
	OneConstDerivatives ComputeDerivatives(std::size_t index, scalar *directors, const std::size_t *neighbors,
		const scalar *dx, const scalar *dy, const scalar *dz, const scalar *lap, std::size_t N, std::size_t Nactive, int k) {

		OneConstDerivatives derivatives;
		int mapoffset = k * index;

		scalar nx, ny, nz;

		std::size_t nbh;
		for (int i = 0; i < k; i++) {

			nbh = neighbors[i * Nactive + index];

			nx = directors[nbh];
			ny = directors[nbh + N];
			nz = directors[nbh + 2 * N];

			derivatives.Dx.nx += dx[mapoffset + i] * nx;
			derivatives.Dx.ny += dx[mapoffset + i] * ny;
			derivatives.Dx.nz += dx[mapoffset + i] * nz;

			derivatives.Dy.nx += dy[mapoffset + i] * nx;
			derivatives.Dy.ny += dy[mapoffset + i] * ny;
			derivatives.Dy.nz += dy[mapoffset + i] * nz;

			derivatives.Dz.nx += dz[mapoffset + i] * nx;
			derivatives.Dz.ny += dz[mapoffset + i] * ny;
			derivatives.Dz.nz += dz[mapoffset + i] * nz;

			derivatives.Dlap.nx += lap[mapoffset + i] * nx;
			derivatives.Dlap.ny += lap[mapoffset + i] * ny;
			derivatives.Dlap.nz += lap[mapoffset + i] * nz;
		}


		return derivatives;
	}

	HEMI_DEV_CALLABLE
	void UpdateDirectorsAlgebraic(std::size_t glob_idx, scalar *directors, scalar lap, std::size_t N, const OneConstDerivatives &d, scalar chir, scalar rate) {

		scalar nx000 = directors[glob_idx];
		scalar ny000 = directors[glob_idx + N];
		scalar nz000 = directors[glob_idx + 2 * N];

		directors[glob_idx] = (1.0 + rate) / lap * (4.0 * PI * chir * (d.Dy.nz - d.Dz.ny) - d.Dlap.nx + lap * nx000) - rate * nx000;
		directors[glob_idx + N] = (1.0 + rate) / lap * (4.0 * PI * chir * (d.Dz.nx - d.Dx.nz) - d.Dlap.ny + lap * ny000) - rate * ny000;
		directors[glob_idx + 2 * N] = (1.0 + rate) / lap * (4.0 * PI * chir * (d.Dx.ny - d.Dy.nx) - d.Dlap.nz + lap * nz000) - rate * nz000;
	}

	HEMI_DEV_CALLABLE
	void Normalize(scalar *nn, std::size_t idx, std::size_t N) {
		scalar nx = nn[idx];
		scalar ny = nn[idx + N];
		scalar nz = nn[idx + N * 2];
		scalar len = sqrt(nx * nx + ny * ny + nz * nz);
		nn[idx] /= len;
		nn[idx + N] /= len;
		nn[idx + N * 2] /= len;
	}

	void OneConstAlgebraic(scalar * directors, const std::size_t * active_nodes, const std::size_t * neighbors, const scalar * dx, const scalar * dy, const scalar * dz, const scalar * lap,
		std::size_t N, std::size_t Nactive, int k, scalar chirality, scalar rate) {
		hemi::parallel_for(0u, Nactive, [=] HEMI_LAMBDA(unsigned int idx) {
			
			OneConstDerivatives derivatives = ComputeDerivatives(idx, directors, neighbors, dx, dy, dz, lap, N, Nactive, k);
			UpdateDirectorsAlgebraic(active_nodes[idx], directors, lap[k * idx], N, derivatives, chirality, rate);
			Normalize(directors, active_nodes[idx], N);

		});

	}



	void RelaxGPUOneConst(scalar* directors, const std::size_t* active_nodes, const std::size_t* neighbors, const scalar* dx, const scalar* dy, const scalar* dz, const scalar* lap,
		std::size_t N, std::size_t Nactive, std::size_t k, scalar chirality, scalar rate, std::size_t iterations) {

		hemi::Array<scalar> A_directors(N * 3);
		hemi::Array<std::size_t> A_active_nodes(Nactive);
		hemi::Array<std::size_t> A_neighbors(Nactive * k);
		hemi::Array<scalar> A_dx(Nactive * k);
		hemi::Array<scalar> A_dy(Nactive * k);
		hemi::Array<scalar> A_dz(Nactive * k);
		hemi::Array<scalar> A_lap(Nactive * k);

		A_directors.copyFromHost(directors, N * 3);
		A_active_nodes.copyFromHost(active_nodes, Nactive);
		A_neighbors.copyFromHost(neighbors, Nactive * k);
		A_dx.copyFromHost(dx, Nactive * k);
		A_dy.copyFromHost(dy, Nactive * k);
		A_dz.copyFromHost(dz, Nactive * k);
		A_lap.copyFromHost(lap, Nactive * k);


		typedef void(*method_t)(scalar*, const std::size_t*, const std::size_t*,
			const scalar*, const scalar*, const scalar*, const scalar*,
			std::size_t, std::size_t, int, scalar, scalar);
		method_t method;

		method = OneConstAlgebraic;


		for (int i = 0; i < iterations; i++) {
			// Call relax function
			method(A_directors.devicePtr(), A_active_nodes.readOnlyDevicePtr(), A_neighbors.readOnlyDevicePtr(),
				A_dx.readOnlyDevicePtr(), A_dy.readOnlyDevicePtr(), A_dz.readOnlyDevicePtr(), A_lap.readOnlyDevicePtr(), N, Nactive, k, chirality, rate);
		}

		hemi::synchronize();

		// Copy data back
		hipMemcpy(directors, A_directors.readOnlyHostPtr(), 3 * sizeof(scalar) * N, hipMemcpyDeviceToHost);
	}

}}}}