#include "hip/hip_runtime.h"
#include "CudaContext.cuh"
#include "scalar.h"


namespace LC { namespace FrankOseen { namespace ElasticOnly { namespace FD {

	typedef void(*vFunction_t)(void* data, unsigned int);

	HEMI_DEV_CALLABLE
	void Normalize_Device(scalar* nn, unsigned int idx, unsigned int N) {
		scalar nx = nn[idx];
		scalar ny = nn[idx + N];
		scalar nz = nn[idx + N * 2];
		scalar len = sqrt(nx * nx + ny * ny + nz * nz);
		nn[idx] /= len;
		nn[idx + N] /= len;
		nn[idx + N * 2] /= len;
	}

	HEMI_DEV_CALLABLE
		void HandleBoundaryConditionsOrder2_Device(scalar* nn, unsigned int idx, const int* vXi, const bool* bc, unsigned int N) {
		using namespace LC::Cuda;

		int r[3];
		ind2sub(idx, vXi, r);

		for (int d = 0; d < 3; d++) {
			if (bc[0] && r[0] == 0) nn[idx + N * d] = nn[sub2ind(vXi[0] - 2, r[1], r[2], vXi) + N * d];
			else if (bc[0] && r[0] == vXi[0] - 1) nn[idx + N * d] = nn[sub2ind(1, r[1], r[2], vXi) + N * d];

			if (bc[1] && r[1] == 0) nn[idx + N * d] = nn[sub2ind(r[0], vXi[1] - 2, r[2], vXi) + N * d];
			else if (bc[1] && r[1] == vXi[1] - 1) nn[idx + N * d] = nn[sub2ind(r[0], 1, r[2], vXi) + N * d];

			if (bc[2] && r[2] == 0) nn[idx + N * d] = nn[sub2ind(r[0], r[1], vXi[2] - 2, vXi) + N * d];
			else if (bc[2] && r[2] == vXi[2] - 1) nn[idx + N * d] = nn[sub2ind(r[0], r[1], 1, vXi) + N * d];
		}
	}

	HEMI_DEV_CALLABLE
		void HandleBoundaryConditionsOrder4_Device(scalar* nn, unsigned int idx, const int* vXi, const bool* bc, unsigned int N) {
		using namespace LC::Cuda;

		int r[3];
		ind2sub(idx, vXi, r);

		for (int d = 0; d < 3; d++) {
			if (bc[0] && r[0] == 0) nn[idx + N * d] = nn[sub2ind(vXi[0] - 4, r[1], r[2], vXi) + N * d];
			else if (bc[0] && r[0] == 1) nn[idx + N * d] = nn[sub2ind(vXi[0] - 3, r[1], r[2], vXi) + N * d];
			else if (bc[0] && r[0] == vXi[0] - 2) nn[idx + N * d] = nn[sub2ind(2, r[1], r[2], vXi) + N * d];
			else if (bc[0] && r[0] == vXi[0] - 1) nn[idx + N * d] = nn[sub2ind(3, r[1], r[2], vXi) + N * d];

			if (bc[1] && r[1] == 0) nn[idx + N * d] = nn[sub2ind(r[0], vXi[1] - 4, r[2], vXi) + N * d];
			else if (bc[1] && r[1] == 1) nn[idx + N * d] = nn[sub2ind(r[0], vXi[1] - 3, r[2], vXi) + N * d];
			else if (bc[1] && r[1] == vXi[1] - 2) nn[idx + N * d] = nn[sub2ind(r[0], 2, r[2], vXi) + N * d];
			else if (bc[1] && r[1] == vXi[1] - 1) nn[idx + N * d] = nn[sub2ind(r[0], 3, r[2], vXi) + N * d];

			if (bc[2] && r[2] == 0) nn[idx + N * d] = nn[sub2ind(r[0], r[1], vXi[2] - 4, vXi) + N * d];
			else if (bc[2] && r[2] == 1) nn[idx + N * d] = nn[sub2ind(r[0], r[1], vXi[2] - 3, vXi) + N * d];
			else if (bc[2] && r[2] == vXi[2] - 2) nn[idx + N * d] = nn[sub2ind(r[0], r[1], 2, vXi) + N * d];
			else if (bc[2] && r[2] == vXi[2] - 1) nn[idx + N * d] = nn[sub2ind(r[0], r[1], 3, vXi) + N * d];
		}
	}


	// Update bulk nodes
	HEMI_DEV_CALLABLE
	void OneConstAlgebraicO2_Device(scalar* nn, unsigned int idx, unsigned int Nd, const int* vXi, const scalar* dr, const scalar *dr2, scalar rate, scalar chirality) {
		using namespace LC::Cuda;


		int r[3];
		ind2sub(idx, vXi, r);

		for (int d = 0; d < 3; d++)
			if (r[d] == 0 || r[d] == vXi[d] - 1) return;

		scalar N, curl;

		scalar nD[3][3];
		// [position][direction][front/back]
		scalar dir[3][3][2];
		scalar vol = 1.0, denom = 0.0;

		for (int d = 0; d < 3; d++) {
			vol *= dr2[d];
			denom += dr2[d] * dr2[(d + 1) % 3];

			// Fill
			dir[0][d][0] = nn[sub2ind(r[0] - 1, r[1], r[2], vXi) + Nd * d];
			dir[0][d][1] = nn[sub2ind(r[0] + 1, r[1], r[2], vXi) + Nd * d];

			dir[1][d][0] = nn[sub2ind(r[0], r[1] - 1, r[2], vXi) + Nd * d];
			dir[1][d][1] = nn[sub2ind(r[0], r[1] + 1, r[2], vXi) + Nd * d];

			dir[2][d][0] = nn[sub2ind(r[0], r[1], r[2] - 1, vXi) + Nd * d];
			dir[2][d][1] = nn[sub2ind(r[0], r[1], r[2] + 1, vXi) + Nd * d];

			nD[0][d] = (dir[0][d][1] - dir[0][d][0]) / (2.0 * dr[0]);
			nD[1][d] = (dir[1][d][1] - dir[1][d][0]) / (2.0 * dr[1]);
			nD[2][d] = (dir[2][d][1] - dir[2][d][0]) / (2.0 * dr[2]);
		}

		__syncthreads();

		scalar c = (1.0 + rate) / (2.0 * denom);

		for (int d = 0; d < 3; d++) {

			N = 0.0;

			for (int dd = 0; dd < 3; dd++)
				N += (vol / dr2[dd]) * (dir[dd][d][1] + dir[dd][d][0]);

			const int a = (d + 1) % 3;
			const int b = (d + 2) % 3;
			curl = nD[a][b] - nD[b][a];

			nn[idx + Nd * d] = c * (N - 4.0 * PI * chirality * vol * curl) - rate * nn[idx + Nd * d];
		}
	}

	HEMI_DEV_CALLABLE
		void OneConstAlgebraicO4_Device(scalar* nn, unsigned int idx, unsigned int Nd, const int* vXi, const scalar* dr, const scalar* dr2, scalar rate, scalar chirality) {
		using namespace LC::Cuda;

		int r[3];
		ind2sub(idx, vXi, r);

		for (int d = 0; d < 3; d++)
			if (r[d] < 2 || r[d] > vXi[d] - 3) return;

		constexpr scalar c1 = 1.0 / 12.0;
		constexpr scalar c2 = 2.0 / 3.0;
		constexpr scalar c3 = 4.0 / 3.0;

		scalar N, curl;
		// [position][director]
		scalar nAvg[3][3];
		// [derivative][director]
		scalar nD[3][3];
		// [position][director][--, -, +, ++]
		scalar dir[3][3][4];

		for (int d = 0; d < 3; d++) {

			// Fill
			dir[0][d][0] = nn[sub2ind(r[0] - 2, r[1], r[2], vXi) + Nd * d];
			dir[0][d][1] = nn[sub2ind(r[0] - 1, r[1], r[2], vXi) + Nd * d];
			dir[0][d][2] = nn[sub2ind(r[0] + 1, r[1], r[2], vXi) + Nd * d];
			dir[0][d][3] = nn[sub2ind(r[0] + 2, r[1], r[2], vXi) + Nd * d];

			dir[1][d][0] = nn[sub2ind(r[0], r[1] - 2, r[2], vXi) + Nd * d];
			dir[1][d][1] = nn[sub2ind(r[0], r[1] - 1, r[2], vXi) + Nd * d];
			dir[1][d][2] = nn[sub2ind(r[0], r[1] + 1, r[2], vXi) + Nd * d];
			dir[1][d][3] = nn[sub2ind(r[0], r[1] + 2, r[2], vXi) + Nd * d];

			dir[2][d][0] = nn[sub2ind(r[0], r[1], r[2] - 2, vXi) + Nd * d];
			dir[2][d][1] = nn[sub2ind(r[0], r[1], r[2] - 1, vXi) + Nd * d];
			dir[2][d][2] = nn[sub2ind(r[0], r[1], r[2] + 1, vXi) + Nd * d];
			dir[2][d][3] = nn[sub2ind(r[0], r[1], r[2] + 2, vXi) + Nd * d];

			for (int i = 0; i < 3; i++) {
				nD[i][d] = (-c1 * dir[i][d][3] + c2 * dir[i][d][2] - c2 * dir[i][d][1] + c1 * dir[i][d][0]) / dr[i];
				nAvg[i][d] = (c3 * (dir[i][d][1] + dir[i][d][2]) - c1 * (dir[i][d][0] + dir[i][d][3])) / dr2[i];
			}

		}

		__syncthreads();

		scalar drinv = 5.0 / 2.0 * (1.0 / dr2[0] + 1.0 / dr2[1] + 1.0 / dr2[2]);

		for (int d = 0; d < 3; d++) {

			N = nAvg[0][d] + nAvg[1][d] + nAvg[2][d];

			const int a = (d + 1) % 3;
			const int b = (d + 2) % 3;
			curl = nD[a][b] - nD[b][a];

			nn[idx + Nd * d] = (1.0 + rate) * (N - 4.0 * PI * chirality * curl) / drinv - rate * nn[idx + Nd * d];
		}
	}


	void OneConstAlgebraicO2(scalar* directors, const int* vXi, const bool* bc, const scalar* cXi, const scalar *dr, const scalar *dr2, scalar chirality, scalar rate, unsigned int N) {
		
		hemi::parallel_for(0u, N, [=] HEMI_LAMBDA(unsigned int idx) {
			OneConstAlgebraicO2_Device(directors, idx, N, vXi, dr, dr2, rate, chirality);
			HandleBoundaryConditionsOrder2_Device(directors, idx, vXi, bc, N);
			Normalize_Device(directors, idx, N);
		});
	}

	void OneConstAlgebraicO4(scalar* directors, const int* vXi, const bool* bc, const scalar* cXi, const scalar* dr, const scalar* dr2, scalar chirality, scalar rate, unsigned int N) {

		hemi::parallel_for(0u, N, [=] HEMI_LAMBDA(unsigned int idx) {
			HandleBoundaryConditionsOrder4_Device(directors, idx, vXi, bc, N);
			OneConstAlgebraicO4_Device(directors, idx, N, vXi, dr, dr2, rate, chirality);
			Normalize_Device(directors, idx, N);
		});
	}

	/* routine
		0 - FullFunctionalO2
		1 - OneConstFunctionalO2
		2 - FullAlgebraicO2
		3 - OneConstAlgebraicO2
		4 - FullFunctionalO4
		5 - OneConstFunctionalO4
		6 - FullAlgebraicO4
		7 - OneConstAlgebraicO4
	*/
	void RelaxGPU(scalar* directors, const int* vXi, const bool* bc, const scalar* cXi, scalar chirality, scalar rate, unsigned int iterations, int routine) {
		unsigned int N = vXi[0] * vXi[1] * vXi[2];

		hemi::Array<scalar> dirs(N * 3);
		hemi::Array<scalar> cX(3);
		hemi::Array<int> vX(3);
		hemi::Array<bool> BC(3);

		dirs.copyFromHost(directors, N * 3);
		cX.copyFromHost(cXi, 3);
		vX.copyFromHost(vXi, 3);
		BC.copyFromHost(bc, 3);

		hemi::Array<scalar> dr(3), dr2(3);
		{
			scalar* h_dr = dr.writeOnlyHostPtr();
			scalar* h_dr2 = dr2.writeOnlyHostPtr();
			for (int d = 0; d < 3; d++) {
				h_dr[d] = cXi[d] / (scalar)(vXi[d] - 1);
				h_dr2[d] = h_dr[d] * h_dr[d];
			}
		}

		// Flipped algebraic bit
		if (routine & 0x02) {
			// Flipped one const bit
			if (routine & 0x01) {
				typedef void(*method_t)(scalar*, const int*, const bool*, const scalar*, const scalar*, const scalar*, scalar, scalar, unsigned int);
				method_t method;
				// Flipped order4 bit
				if (routine & 0x04) method = OneConstAlgebraicO4;
				else method = OneConstAlgebraicO2;

				for (unsigned int i = 0; i < iterations; i++)
					method(dirs.devicePtr(),
						vX.readOnlyDevicePtr(),
						BC.readOnlyDevicePtr(),
						cX.readOnlyDevicePtr(),
						dr.readOnlyDevicePtr(),
						dr2.readOnlyDevicePtr(),
						chirality, rate, N);
			}
			else {
				return;
			}

		}
		else {
			return;
		}
		hemi::synchronize();
		hipMemcpy(directors, dirs.readOnlyHostPtr(), 3 * sizeof(scalar) * N, hipMemcpyDeviceToHost);
	}


}}

namespace Electric { namespace FD {

	typedef void(*vFunction_t)(void* data, unsigned int);

	HEMI_DEV_CALLABLE
		void Normalize_Device(scalar* nn, unsigned int idx, unsigned int N) {
		scalar nx = nn[idx];
		scalar ny = nn[idx + N];
		scalar nz = nn[idx + N * 2];
		scalar len = sqrt(nx * nx + ny * ny + nz * nz);
		nn[idx] /= len;
		nn[idx + N] /= len;
		nn[idx + N * 2] /= len;
	}

	HEMI_DEV_CALLABLE
		void HandleBoundaryConditionsOrder2_Device(scalar* nn, scalar *vv, unsigned int idx, const int* vXi, const bool* bc, unsigned int N) {
		using namespace LC::Cuda;

		int r[3];
		ind2sub(idx, vXi, r);

		for (int d = 0; d < 3; d++) {
			if (bc[0] && r[0] == 0) nn[idx + N * d] = nn[sub2ind(vXi[0] - 2, r[1], r[2], vXi) + N * d];
			else if (bc[0] && r[0] == vXi[0] - 1) nn[idx + N * d] = nn[sub2ind(1, r[1], r[2], vXi) + N * d];

			if (bc[1] && r[1] == 0) nn[idx + N * d] = nn[sub2ind(r[0], vXi[1] - 2, r[2], vXi) + N * d];
			else if (bc[1] && r[1] == vXi[1] - 1) nn[idx + N * d] = nn[sub2ind(r[0], 1, r[2], vXi) + N * d];

			if (bc[2] && r[2] == 0) nn[idx + N * d] = nn[sub2ind(r[0], r[1], vXi[2] - 2, vXi) + N * d];
			else if (bc[2] && r[2] == vXi[2] - 1) nn[idx + N * d] = nn[sub2ind(r[0], r[1], 1, vXi) + N * d];
		}

		if (bc[0] && r[0] == 0) vv[idx] = vv[sub2ind(vXi[0] - 2, r[1], r[2], vXi)];
		else if (bc[0] && r[0] == vXi[0] - 1) vv[idx] = vv[sub2ind(1, r[1], r[2], vXi)];

		if (bc[1] && r[1] == 0) vv[idx] = vv[sub2ind(r[0], vXi[1] - 2, r[2], vXi)];
		else if (bc[1] && r[1] == vXi[1] - 1) nn[idx] = vv[sub2ind(r[0], 1, r[2], vXi)];

		if (bc[2] && r[2] == 0) vv[idx] = vv[sub2ind(r[0], r[1], vXi[2] - 2, vXi)];
		else if (bc[2] && r[2] == vXi[2] - 1) vv[idx] = vv[sub2ind(r[0], r[1], 1, vXi)];
	}

	HEMI_DEV_CALLABLE
		void HandleBoundaryConditionsOrder4_Device(scalar* nn, scalar *vv, unsigned int idx, const int* vXi, const bool* bc, unsigned int N) {
		using namespace LC::Cuda;

		int r[3];
		ind2sub(idx, vXi, r);

		for (int d = 0; d < 3; d++) {
			if (bc[0] && r[0] == 0) nn[idx + N * d] = nn[sub2ind(vXi[0] - 4, r[1], r[2], vXi) + N * d];
			else if (bc[0] && r[0] == 1) nn[idx + N * d] = nn[sub2ind(vXi[0] - 3, r[1], r[2], vXi) + N * d];
			else if (bc[0] && r[0] == vXi[0] - 2) nn[idx + N * d] = nn[sub2ind(2, r[1], r[2], vXi) + N * d];
			else if (bc[0] && r[0] == vXi[0] - 1) nn[idx + N * d] = nn[sub2ind(3, r[1], r[2], vXi) + N * d];

			if (bc[1] && r[1] == 0) nn[idx + N * d] = nn[sub2ind(r[0], vXi[1] - 4, r[2], vXi) + N * d];
			else if (bc[1] && r[1] == 1) nn[idx + N * d] = nn[sub2ind(r[0], vXi[1] - 3, r[2], vXi) + N * d];
			else if (bc[1] && r[1] == vXi[1] - 2) nn[idx + N * d] = nn[sub2ind(r[0], 2, r[2], vXi) + N * d];
			else if (bc[1] && r[1] == vXi[1] - 1) nn[idx + N * d] = nn[sub2ind(r[0], 3, r[2], vXi) + N * d];

			if (bc[2] && r[2] == 0) nn[idx + N * d] = nn[sub2ind(r[0], r[1], vXi[2] - 4, vXi) + N * d];
			else if (bc[2] && r[2] == 1) nn[idx + N * d] = nn[sub2ind(r[0], r[1], vXi[2] - 3, vXi) + N * d];
			else if (bc[2] && r[2] == vXi[2] - 2) nn[idx + N * d] = nn[sub2ind(r[0], r[1], 2, vXi) + N * d];
			else if (bc[2] && r[2] == vXi[2] - 1) nn[idx + N * d] = nn[sub2ind(r[0], r[1], 3, vXi) + N * d];
		}

		if (bc[0] && r[0] == 0) vv[idx] = vv[sub2ind(vXi[0] - 4, r[1], r[2], vXi)];
		else if (bc[0] && r[0] == 1) vv[idx] = vv[sub2ind(vXi[0] - 3, r[1], r[2], vXi)];
		else if (bc[0] && r[0] == vXi[0] - 2) vv[idx] = vv[sub2ind(2, r[1], r[2], vXi)];
		else if (bc[0] && r[0] == vXi[0] - 1) vv[idx] = vv[sub2ind(3, r[1], r[2], vXi)];

		if (bc[1] && r[1] == 0) vv[idx] = vv[sub2ind(r[0], vXi[1] - 4, r[2], vXi)];
		else if (bc[1] && r[1] == 1) vv[idx] = vv[sub2ind(r[0], vXi[1] - 3, r[2], vXi)];
		else if (bc[1] && r[1] == vXi[1] - 2) vv[idx] = vv[sub2ind(r[0], 2, r[2], vXi)];
		else if (bc[1] && r[1] == vXi[1] - 1) vv[idx] = vv[sub2ind(r[0], 3, r[2], vXi)];

		if (bc[2] && r[2] == 0) vv[idx] = vv[sub2ind(r[0], r[1], vXi[2] - 4, vXi)];
		else if (bc[2] && r[2] == 1) vv[idx] = vv[sub2ind(r[0], r[1], vXi[2] - 3, vXi)];
		else if (bc[2] && r[2] == vXi[2] - 2) vv[idx] = vv[sub2ind(r[0], r[1], 2, vXi)];
		else if (bc[2] && r[2] == vXi[2] - 1) vv[idx] = vv[sub2ind(r[0], r[1], 3, vXi)];
	}


	// Update bulk nodes
	HEMI_DEV_CALLABLE
		void OneConstAlgebraicO2_Device(scalar* nn, scalar *vv, unsigned int idx, unsigned int Nd, const int* vXi, scalar K, scalar epar, scalar eper, const scalar* dr, const scalar* dr2, scalar rate, scalar chirality) {
		using namespace LC::Cuda;


		int r[3];
		ind2sub(idx, vXi, r);

		for (int d = 0; d < 3; d++)
			if (r[d] == 0 || r[d] == vXi[d] - 1) return;

		scalar N, curl;

		scalar nD[3][3];
		scalar vD[3];
		// [position][direction][front/back]
		scalar dir[3][3][2];
		scalar vol = 1.0, denom = 0.0;

		for (int d = 0; d < 3; d++) {
			vol *= dr2[d];
			denom += dr2[d] * dr2[(d + 1) % 3];

			// Fill
			dir[0][d][0] = nn[sub2ind(r[0] - 1, r[1], r[2], vXi) + Nd * d];
			dir[0][d][1] = nn[sub2ind(r[0] + 1, r[1], r[2], vXi) + Nd * d];

			dir[1][d][0] = nn[sub2ind(r[0], r[1] - 1, r[2], vXi) + Nd * d];
			dir[1][d][1] = nn[sub2ind(r[0], r[1] + 1, r[2], vXi) + Nd * d];

			dir[2][d][0] = nn[sub2ind(r[0], r[1], r[2] - 1, vXi) + Nd * d];
			dir[2][d][1] = nn[sub2ind(r[0], r[1], r[2] + 1, vXi) + Nd * d];

			nD[0][d] = (dir[0][d][1] - dir[0][d][0]) / (2.0 * dr[0]);
			nD[1][d] = (dir[1][d][1] - dir[1][d][0]) / (2.0 * dr[1]);
			nD[2][d] = (dir[2][d][1] - dir[2][d][0]) / (2.0 * dr[2]);
		}

		vD[0] = (vv[sub2ind(r[0] + 1, r[1], r[2], vXi)] - vv[sub2ind(r[0] - 1, r[1], r[2], vXi)]) / (2.0 * dr[0]);
		vD[1] = (vv[sub2ind(r[0], r[1] + 1, r[2], vXi)] - vv[sub2ind(r[0], r[1] - 1, r[2], vXi)]) / (2.0 * dr[1]);
		vD[2] = (vv[sub2ind(r[0], r[1], r[2] + 1, vXi)] - vv[sub2ind(r[0], r[1], r[2] - 1, vXi)]) / (2.0 * dr[2]);


		__syncthreads();


		scalar Xi = 8.854 * (epar - eper) / K;

		for (int d = 0; d < 3; d++) {

			scalar c = (1.0 + rate) / (2.0 * denom - dr2[0]* dr2[1]* dr2[2] * Xi * vD[d] * vD[d]);
			N = 0.0;

			for (int dd = 0; dd < 3; dd++)
				N += (vol / dr2[dd]) * (dir[dd][d][1] + dir[dd][d][0]);

			const int a = (d + 1) % 3;
			const int b = (d + 2) % 3;
			curl = nD[a][b] - nD[b][a];

			nn[idx + Nd * d] = c * (N - 4.0 * PI * chirality * vol * curl + Xi * vol * vD[d] * (vD[a] * nn[sub2ind(r[0], r[1], r[2], vXi) + Nd * a] + vD[b] * nn[sub2ind(r[0], r[1], r[2], vXi) + Nd * b])) - rate * nn[idx + Nd * d];
		}
	}

	// Update bulk nodes
	HEMI_DEV_CALLABLE
		void UpdateVoltageO2_Device(scalar* nn, scalar* vv, unsigned int idx, unsigned int Nd, const int* vXi, scalar epar, scalar eper, const scalar* dr, scalar rate) {
		using namespace LC::Cuda;


		int r[3];
		ind2sub(idx, vXi, r);

		for (int d = 0; d < 3; d++)
			if (r[d] == 0 || r[d] == vXi[d] - 1) return;


		scalar nx000 = nn[idx];
		scalar ny000 = nn[idx + Nd];
		scalar nz000 = nn[idx + 2 * Nd];
		scalar ea = epar - eper;

		scalar nx100 = (nn[sub2ind(r[0] + 1, r[1], r[2], vXi)] - nn[sub2ind(r[0] + 1, r[1], r[2], vXi)]) / (2.0 * dr[0]);
		scalar ny100 = (nn[sub2ind(r[0] + 1, r[1], r[2], vXi) + Nd] - nn[sub2ind(r[0] + 1, r[1], r[2], vXi) + Nd]) / (2.0 * dr[0]);
		scalar nz100 = (nn[sub2ind(r[0] + 1, r[1], r[2], vXi) + 2 * Nd] - nn[sub2ind(r[0] + 1, r[1], r[2], vXi) + 2 * Nd]) / (2.0 * dr[0]);

		scalar nx010 = (nn[sub2ind(r[0], r[1] + 1, r[2], vXi)] - nn[sub2ind(r[0], r[1] - 1, r[2], vXi)]) / (2.0 * dr[1]);
		scalar ny010 = (nn[sub2ind(r[0], r[1] + 1, r[2], vXi) + Nd] - nn[sub2ind(r[0], r[1] - 1, r[2], vXi) + Nd]) / (2.0 * dr[1]);
		scalar nz010 = (nn[sub2ind(r[0], r[1] + 1, r[2], vXi) + 2 * Nd] - nn[sub2ind(r[0], r[1] - 1, r[2], vXi) + 2 * Nd]) / (2.0 * dr[1]);

		scalar nx001 = (nn[sub2ind(r[0], r[1], r[2] + 1, vXi)] - nn[sub2ind(r[0], r[1], r[2] - 1, vXi)]) / (2.0 * dr[2]);
		scalar ny001 = (nn[sub2ind(r[0], r[1], r[2] + 1, vXi) + Nd] - nn[sub2ind(r[0], r[1], r[2] - 1, vXi) + Nd]) / (2.0 * dr[2]);
		scalar nz001 = (nn[sub2ind(r[0], r[1], r[2] + 1, vXi) + 2 * Nd] - nn[sub2ind(r[0], r[1], r[2] - 1, vXi) + 2 * Nd]) / (2.0 * dr[2]);


		scalar v100 = (vv[sub2ind(r[0] + 1, r[1], r[2], vXi)] - vv[sub2ind(r[0] - 1, r[1], r[2], vXi)]) / (2.0 * dr[0]);
		scalar v010 = (vv[sub2ind(r[0], r[1] + 1, r[2], vXi)] - vv[sub2ind(r[0], r[1] - 1, r[2], vXi)]) / (2.0 * dr[1]);
		scalar v001 = (vv[sub2ind(r[0], r[1], r[2] + 1, vXi)] - vv[sub2ind(r[0], r[1], r[2] - 1, vXi)]) / (2.0 * dr[2]);

		scalar v110 = (vv[sub2ind(r[0] + 1, r[1] + 1, r[2], vXi)] - vv[sub2ind(r[0] + 1, r[1] - 1, r[2], vXi)] - vv[sub2ind(r[0] - 1, r[1] + 1, r[2], vXi)] + vv[sub2ind(r[0] - 1, r[1] - 1, r[2], vXi)]) / (4.0 * dr[0] * dr[1]);
		scalar v101 = (vv[sub2ind(r[0] + 1, r[1], r[2] + 1, vXi)] - vv[sub2ind(r[0] + 1, r[1], r[2] - 1, vXi)] - vv[sub2ind(r[0] - 1, r[1], r[2] + 1, vXi)] + vv[sub2ind(r[0] - 1, r[1], r[2] - 1, vXi)]) / (4.0 * dr[0] * dr[2]);
		scalar v011 = (vv[sub2ind(r[0], r[1] + 1, r[2] + 1, vXi)] - vv[sub2ind(r[0], r[1] + 1, r[2] - 1, vXi)] - vv[sub2ind(r[0], r[1] - 1, r[2] + 1, vXi)] + vv[sub2ind(r[0], r[1] - 1, r[2] - 1, vXi)]) / (4.0 * dr[1] * dr[2]);

		scalar w200 = -2.0 / (dr[0] * dr[0]);
		scalar vm200 = (vv[sub2ind(r[0] + 1, r[1], r[2], vXi)] + vv[sub2ind(r[0] + 1, r[1], r[2], vXi)]) / (dr[0] * dr[0]);

		scalar w020 = -2.0 / (dr[1] * dr[1]);
		scalar vm020 = (vv[sub2ind(r[0], r[1] + 1, r[2], vXi)] + vv[sub2ind(r[0], r[1] + 1, r[2], vXi)]) / (dr[1] * dr[1]);

		scalar w002 = -2.0 / (dr[2] * dr[2]);
		scalar vm002 = (vv[sub2ind(r[0], r[1], r[2] + 1, vXi)] + vv[sub2ind(r[0], r[1], r[2] + 1, vXi)]) / (dr[2] * dr[2]);


		__syncthreads();

		vv[idx] = (1. + rate) * (-9. * ea * (-3. * (vm002 + vm020 + vm200) + nx100 * (ny000 * v010 + nz000 * v001 + 2. * nx000 * v100) +
			(nx000 * nx000) * vm200 + (ny000 * ny000) * vm020 + (nz000 * nz000) * vm002 + nx000 * ny010 * v100 + nx000 * ny100 * v010 +
			nx000 * nz001 * v100 + nx000 * nz100 * v001 + nx001 * nz000 * v100 + nx010 * ny000 * v100 + ny000 * nz001 * v010 + ny000 * nz010 * v001 +
			ny001 * nz000 * v010 + ny010 * nz000 * v001 + 2. * nx000 * ny000 * v110 + 2. * nx000 * nz000 * v101 + 2. * ny000 * ny010 * v010 + 2. * ny000 * nz000 * v011 +
			2. * nz000 * nz001 * v001) - 2. * (epar + 2. * eper) * (vm002 + vm020 + vm200)) / (2. * (epar + 2. * eper) * (w002 + w020 + w200) +
				9. * ea * ((-3. + nx000 * nx000) * w200 + (-3. + ny000 * ny000) * w020 + (-3. + nz000 * nz000) * w002)) - rate * vv[idx];
		
	}

	HEMI_DEV_CALLABLE
		void OneConstAlgebraicO4_Device(scalar* nn, scalar*vv, unsigned int idx, unsigned int Nd, const int* vXi, scalar K, scalar epar, scalar eper, const scalar* dr, const scalar* dr2, scalar rate, scalar chirality) {
		using namespace LC::Cuda;

		int r[3];
		ind2sub(idx, vXi, r);

		for (int d = 0; d < 3; d++)
			if (r[d] < 2 || r[d] > vXi[d] - 3) return;

		constexpr scalar c1 = 1.0 / 12.0;
		constexpr scalar c2 = 2.0 / 3.0;
		constexpr scalar c3 = 4.0 / 3.0;

		scalar N, curl;
		// [position][director]
		scalar nAvg[3][3];
		scalar vD[3];
		// [derivative][director]
		scalar nD[3][3];
		// [derivative][director][--, -, +, ++]
		scalar dir[3][3][4];

		for (int d = 0; d < 3; d++) {

			// Fill
			dir[0][d][0] = nn[sub2ind(r[0] - 2, r[1], r[2], vXi) + Nd * d];
			dir[0][d][1] = nn[sub2ind(r[0] - 1, r[1], r[2], vXi) + Nd * d];
			dir[0][d][2] = nn[sub2ind(r[0] + 1, r[1], r[2], vXi) + Nd * d];
			dir[0][d][3] = nn[sub2ind(r[0] + 2, r[1], r[2], vXi) + Nd * d];

			dir[1][d][0] = nn[sub2ind(r[0], r[1] - 2, r[2], vXi) + Nd * d];
			dir[1][d][1] = nn[sub2ind(r[0], r[1] - 1, r[2], vXi) + Nd * d];
			dir[1][d][2] = nn[sub2ind(r[0], r[1] + 1, r[2], vXi) + Nd * d];
			dir[1][d][3] = nn[sub2ind(r[0], r[1] + 2, r[2], vXi) + Nd * d];

			dir[2][d][0] = nn[sub2ind(r[0], r[1], r[2] - 2, vXi) + Nd * d];
			dir[2][d][1] = nn[sub2ind(r[0], r[1], r[2] - 1, vXi) + Nd * d];
			dir[2][d][2] = nn[sub2ind(r[0], r[1], r[2] + 1, vXi) + Nd * d];
			dir[2][d][3] = nn[sub2ind(r[0], r[1], r[2] + 2, vXi) + Nd * d];

			for (int i = 0; i < 3; i++) {
				nD[i][d] = (-c1 * dir[i][d][3] + c2 * dir[i][d][2] - c2 * dir[i][d][1] + c1 * dir[i][d][0]) / dr[i];
				nAvg[i][d] = (c3 * (dir[i][d][1] + dir[i][d][2]) - c1 * (dir[i][d][0] + dir[i][d][3])) / dr2[i];
			}

		}

		vD[0] = (-c1 * vv[sub2ind(r[0] + 2, r[1], r[2], vXi)] + c2 * vv[sub2ind(r[0] + 1, r[1], r[2], vXi)] - c2 * vv[sub2ind(r[0] - 1, r[1], r[2], vXi)] + c1 * vv[sub2ind(r[0] - 2, r[1], r[2], vXi)]) / dr[0];
		vD[1] = (-c1 * vv[sub2ind(r[0], r[1] + 2, r[2], vXi)] + c2 * vv[sub2ind(r[0], r[1] + 1, r[2], vXi)] - c2 * vv[sub2ind(r[0], r[1] - 1, r[2], vXi)] + c1 * vv[sub2ind(r[0], r[1] - 2, r[2], vXi)]) / dr[1];
		vD[2] = (-c1 * vv[sub2ind(r[0], r[1], r[2] + 2, vXi)] + c2 * vv[sub2ind(r[0], r[1], r[2] + 1, vXi)] - c2 * vv[sub2ind(r[0], r[1], r[2] - 1, vXi)] + c1 * vv[sub2ind(r[0], r[1], r[2] - 2, vXi)]) / dr[2];
		
		__syncthreads();

		scalar Xi = 8.854 * (epar - eper) / K;

		for (int d = 0; d < 3; d++) {

			N = nAvg[0][d] + nAvg[1][d] + nAvg[2][d];
			scalar denom = 2.5 * (1.0 / dr2[0] + 1.0 / dr2[1] + 1.0 / dr2[2]) - Xi * vD[d] * vD[d];

			const int a = (d + 1) % 3;
			const int b = (d + 2) % 3;
			curl = nD[a][b] - nD[b][a];

			scalar na = nn[sub2ind(r[0], r[1], r[2], vXi) + Nd * a];
			scalar nb = nn[sub2ind(r[0], r[1], r[2], vXi) + Nd * b];


			nn[idx + Nd * d] = (1.0 + rate) * (N - 4.0 * PI * chirality * curl + Xi * vD[d] * (vD[a] * na + vD[b] * nb)) / denom - rate * nn[idx + Nd * d];
		}
	}

	HEMI_DEV_CALLABLE
		void ThreeConstAlgebraicO4_Device(scalar* nn, scalar* vv, unsigned int idx, unsigned int Nd, const int* vXi, scalar k11, scalar k22, scalar k33, scalar epar, scalar eper, const scalar* dr, const scalar* dr2, scalar rate, scalar chirality) {
		using namespace LC::Cuda;

		int r[3];
		ind2sub(idx, vXi, r);

		for (int d = 0; d < 3; d++)
			if (r[d] < 2 || r[d] > vXi[d] - 3) return;

		constexpr scalar c1 = 1.0 / 12.0;
		constexpr scalar c2 = 2.0 / 3.0;
		constexpr scalar c3 = 4.0 / 3.0;

		// [position][director]
		scalar nAvg[3][3];
		scalar v100, v010, v001;
		// [derivative][director]
		scalar nD[3][3];
		// [derivative][director][--, -, +, ++]
		scalar dir[3][3][4];

		scalar nx000 = nn[sub2ind(r[0], r[1], r[2], vXi)];
		scalar ny000 = nn[sub2ind(r[0], r[1], r[2], vXi) + Nd];
		scalar nz000 = nn[sub2ind(r[0], r[1], r[2], vXi) + Nd * 2];

		for (int d = 0; d < 3; d++) {

			// Fill
			dir[0][d][0] = nn[sub2ind(r[0] - 2, r[1], r[2], vXi) + Nd * d];
			dir[0][d][1] = nn[sub2ind(r[0] - 1, r[1], r[2], vXi) + Nd * d];
			dir[0][d][2] = nn[sub2ind(r[0] + 1, r[1], r[2], vXi) + Nd * d];
			dir[0][d][3] = nn[sub2ind(r[0] + 2, r[1], r[2], vXi) + Nd * d];

			dir[1][d][0] = nn[sub2ind(r[0], r[1] - 2, r[2], vXi) + Nd * d];
			dir[1][d][1] = nn[sub2ind(r[0], r[1] - 1, r[2], vXi) + Nd * d];
			dir[1][d][2] = nn[sub2ind(r[0], r[1] + 1, r[2], vXi) + Nd * d];
			dir[1][d][3] = nn[sub2ind(r[0], r[1] + 2, r[2], vXi) + Nd * d];

			dir[2][d][0] = nn[sub2ind(r[0], r[1], r[2] - 2, vXi) + Nd * d];
			dir[2][d][1] = nn[sub2ind(r[0], r[1], r[2] - 1, vXi) + Nd * d];
			dir[2][d][2] = nn[sub2ind(r[0], r[1], r[2] + 1, vXi) + Nd * d];
			dir[2][d][3] = nn[sub2ind(r[0], r[1], r[2] + 2, vXi) + Nd * d];

			for (int i = 0; i < 3; i++) {
				nD[i][d] = (-c1 * dir[i][d][3] + c2 * dir[i][d][2] - c2 * dir[i][d][1] + c1 * dir[i][d][0]) / dr[i];
				nAvg[i][d] = (c3 * (dir[i][d][1] + dir[i][d][2]) - c1 * (dir[i][d][0] + dir[i][d][3]));
			}

		}


		scalar nx110 = (nn[sub2ind(r[0] - 2, r[1] - 2, r[2], vXi)] + nn[sub2ind(r[0] + 2, r[1] + 2, r[2], vXi)] - nn[sub2ind(r[0] - 2, r[1] + 2, r[2], vXi)] - nn[sub2ind(r[0] + 2, r[1] - 2, r[2], vXi)] +
			(nn[sub2ind(r[0] + 1, r[1] - 2, r[2], vXi)] + nn[sub2ind(r[0] - 2, r[1] + 1, r[2], vXi)] - nn[sub2ind(r[0] - 1, r[1] - 2, r[2], vXi)] - nn[sub2ind(r[0] - 2, r[1] - 1, r[2], vXi)] +
				nn[sub2ind(r[0] + 2, r[1] - 1, r[2], vXi)] + nn[sub2ind(r[0] - 1, r[1] + 2, r[2], vXi)] - nn[sub2ind(r[0] + 1, r[1] + 2, r[2], vXi)] - nn[sub2ind(r[0] + 2, r[1] + 1, r[2], vXi)]) * 8.0f +
			(nn[sub2ind(r[0] + 1, r[1] + 1, r[2], vXi)] + nn[sub2ind(r[0] - 1, r[1] - 1, r[2], vXi)] - nn[sub2ind(r[0] + 1, r[1] - 1, r[2], vXi)] - nn[sub2ind(r[0] - 1, r[1] + 1, r[2], vXi)]) * 64.0f) / (144.0f * dr[0] * dr[1]);

		scalar nx101 = (nn[sub2ind(r[0] - 2, r[1], r[2] - 2, vXi)] + nn[sub2ind(r[0] + 2, r[1], r[2] + 2, vXi)] - nn[sub2ind(r[0] - 2, r[1], r[2] + 2, vXi)] - nn[sub2ind(r[0] + 2, r[1], r[2] - 2, vXi)] +
			(nn[sub2ind(r[0] + 1, r[1], r[2] - 2, vXi)] + nn[sub2ind(r[0] - 2, r[1], r[2] + 1, vXi)] - nn[sub2ind(r[0] - 1, r[1], r[2] - 2, vXi)] - nn[sub2ind(r[0] - 2, r[1], r[2] - 1, vXi)] +
				nn[sub2ind(r[0] + 2, r[1], r[2] - 1, vXi)] + nn[sub2ind(r[0] - 1, r[1], r[2] + 2, vXi)] - nn[sub2ind(r[0] + 1, r[1], r[2] + 2, vXi)] - nn[sub2ind(r[0] + 2, r[1], r[2] + 1, vXi)]) * 8.0f +
			(nn[sub2ind(r[0] + 1, r[1], r[2] + 1, vXi)] + nn[sub2ind(r[0] - 1, r[1], r[2] - 1, vXi)] - nn[sub2ind(r[0] + 1, r[1], r[2] - 1, vXi)] - nn[sub2ind(r[0] - 1, r[1], r[2] + 1, vXi)]) * 64.0f) / (144.0f * dr[0] * dr[2]);

		scalar nx011 = (nn[sub2ind(r[0], r[1] - 2, r[2] - 2, vXi)] + nn[sub2ind(r[0], r[1] + 2, r[2] + 2, vXi)] - nn[sub2ind(r[0], r[1] - 2, r[2] + 2, vXi)] - nn[sub2ind(r[0], r[1] + 2, r[2] - 2, vXi)] +
			(nn[sub2ind(r[0], r[1] + 1, r[2] - 2, vXi)] + nn[sub2ind(r[0], r[1] - 2, r[2] + 1, vXi)] - nn[sub2ind(r[0], r[1] - 1, r[2] - 2, vXi)] - nn[sub2ind(r[0], r[1] - 2, r[2] - 1, vXi)] +
				nn[sub2ind(r[0], r[1] + 2, r[2] - 1, vXi)] + nn[sub2ind(r[0], r[1] - 1, r[2] + 2, vXi)] - nn[sub2ind(r[0], r[1] + 1, r[2] + 2, vXi)] - nn[sub2ind(r[0], r[1] + 2, r[2] + 1, vXi)]) * 8.0f +
			(nn[sub2ind(r[0], r[1] + 1, r[2] + 1, vXi)] + nn[sub2ind(r[0], r[1] - 1, r[2] - 1, vXi)] - nn[sub2ind(r[0], r[1] + 1, r[2] - 1, vXi)] - nn[sub2ind(r[0], r[1] - 1, r[2] + 1, vXi)]) * 64.0f) / (144.0f * dr[1] * dr[2]);


		scalar ny110 = (nn[sub2ind(r[0] - 2, r[1] - 2, r[2], vXi) + Nd] + nn[sub2ind(r[0] + 2, r[1] + 2, r[2], vXi) + Nd] - nn[sub2ind(r[0] - 2, r[1] + 2, r[2], vXi) + Nd] - nn[sub2ind(r[0] + 2, r[1] - 2, r[2], vXi) + Nd] +
			(nn[sub2ind(r[0] + 1, r[1] - 2, r[2], vXi) + Nd] + nn[sub2ind(r[0] - 2, r[1] + 1, r[2], vXi) + Nd] - nn[sub2ind(r[0] - 1, r[1] - 2, r[2], vXi) + Nd] - nn[sub2ind(r[0] - 2, r[1] - 1, r[2], vXi) + Nd] +
				nn[sub2ind(r[0] + 2, r[1] - 1, r[2], vXi) + Nd] + nn[sub2ind(r[0] - 1, r[1] + 2, r[2], vXi) + Nd] - nn[sub2ind(r[0] + 1, r[1] + 2, r[2], vXi) + Nd] - nn[sub2ind(r[0] + 2, r[1] + 1, r[2], vXi) + Nd]) * 8.0f +
			(nn[sub2ind(r[0] + 1, r[1] + 1, r[2], vXi) + Nd] + nn[sub2ind(r[0] - 1, r[1] - 1, r[2], vXi) + Nd] - nn[sub2ind(r[0] + 1, r[1] - 1, r[2], vXi) + Nd] - nn[sub2ind(r[0] - 1, r[1] + 1, r[2], vXi) + Nd]) * 64.0f) / (144.0f * dr[0] * dr[1]);


		scalar ny101 = (nn[sub2ind(r[0] - 2, r[1], r[2] - 2, vXi) + Nd] + nn[sub2ind(r[0] + 2, r[1], r[2] + 2, vXi) + Nd] - nn[sub2ind(r[0] - 2, r[1], r[2] + 2, vXi) + Nd] - nn[sub2ind(r[0] + 2, r[1], r[2] - 2, vXi) + Nd] +
			(nn[sub2ind(r[0] + 1, r[1], r[2] - 2, vXi) + Nd] + nn[sub2ind(r[0] - 2, r[1], r[2] + 1, vXi) + Nd] - nn[sub2ind(r[0] - 1, r[1], r[2] - 2, vXi) + Nd] - nn[sub2ind(r[0] - 2, r[1], r[2] - 1, vXi) + Nd] +
				nn[sub2ind(r[0] + 2, r[1], r[2] - 1, vXi) + Nd] + nn[sub2ind(r[0] - 1, r[1], r[2] + 2, vXi) + Nd] - nn[sub2ind(r[0] + 1, r[1], r[2] + 2, vXi) + Nd] - nn[sub2ind(r[0] + 2, r[1], r[2] + 1, vXi) + Nd]) * 8.0f +
			(nn[sub2ind(r[0] + 1, r[1], r[2] + 1, vXi) + Nd] + nn[sub2ind(r[0] - 1, r[1], r[2] - 1, vXi) + Nd] - nn[sub2ind(r[0] + 1, r[1], r[2] - 1, vXi) + Nd] - nn[sub2ind(r[0] - 1, r[1], r[2] + 1, vXi) + Nd]) * 64.0f) / (144.0f * dr[0] * dr[2]);

		scalar ny011 = (nn[sub2ind(r[0], r[1] - 2, r[2] - 2, vXi) + Nd] + nn[sub2ind(r[0], r[1] + 2, r[2] + 2, vXi) + Nd] - nn[sub2ind(r[0], r[1] - 2, r[2] + 2, vXi) + Nd] - nn[sub2ind(r[0], r[1] + 2, r[2] - 2, vXi) + Nd] +
			(nn[sub2ind(r[0], r[1] + 1, r[2] - 2, vXi) + Nd] + nn[sub2ind(r[0], r[1] - 2, r[2] + 1, vXi) + Nd] - nn[sub2ind(r[0], r[1] - 1, r[2] - 2, vXi) + Nd] - nn[sub2ind(r[0], r[1] - 2, r[2] - 1, vXi) + Nd] +
				nn[sub2ind(r[0], r[1] + 2, r[2] - 1, vXi) + Nd] + nn[sub2ind(r[0], r[1] - 1, r[2] + 2, vXi) + Nd] - nn[sub2ind(r[0], r[1] + 1, r[2] + 2, vXi) + Nd] - nn[sub2ind(r[0], r[1] + 2, r[2] + 1, vXi) + Nd]) * 8.0f +
			(nn[sub2ind(r[0], r[1] + 1, r[2] + 1, vXi) + Nd] + nn[sub2ind(r[0], r[1] - 1, r[2] - 1, vXi) + Nd] - nn[sub2ind(r[0], r[1] + 1, r[2] - 1, vXi) + Nd] - nn[sub2ind(r[0], r[1] - 1, r[2] + 1, vXi) + Nd]) * 64.0f) / (144.0f * dr[1] * dr[2]);

		scalar nz110 = (nn[sub2ind(r[0] - 2, r[1] - 2, r[2], vXi) + 2 * Nd] + nn[sub2ind(r[0] + 2, r[1] + 2, r[2], vXi) + 2 * Nd] - nn[sub2ind(r[0] - 2, r[1] + 2, r[2], vXi) + 2 * Nd] - nn[sub2ind(r[0] + 2, r[1] - 2, r[2], vXi) + 2 * Nd] +
			(nn[sub2ind(r[0] + 1, r[1] - 2, r[2], vXi) + 2 * Nd] + nn[sub2ind(r[0] - 2, r[1] + 1, r[2], vXi) + 2 * Nd] - nn[sub2ind(r[0] - 1, r[1] - 2, r[2], vXi) + 2 * Nd] - nn[sub2ind(r[0] - 2, r[1] - 1, r[2], vXi) + 2 * Nd] +
				nn[sub2ind(r[0] + 2, r[1] - 1, r[2], vXi) + 2 * Nd] + nn[sub2ind(r[0] - 1, r[1] + 2, r[2], vXi) + 2 * Nd] - nn[sub2ind(r[0] + 1, r[1] + 2, r[2], vXi) + 2 * Nd] - nn[sub2ind(r[0] + 2, r[1] + 1, r[2], vXi) + 2 * Nd]) * 8.0f +
			(nn[sub2ind(r[0] + 1, r[1] + 1, r[2], vXi) + 2 * Nd] + nn[sub2ind(r[0] - 1, r[1] - 1, r[2], vXi) + 2 * Nd] - nn[sub2ind(r[0] + 1, r[1] - 1, r[2], vXi) + 2 * Nd] - nn[sub2ind(r[0] - 1, r[1] + 1, r[2], vXi) + 2 * Nd]) * 64.0f) / (144.0f * dr[0] * dr[1]);


		scalar nz101 = (nn[sub2ind(r[0] - 2, r[1], r[2] - 2, vXi) + 2 * Nd] + nn[sub2ind(r[0] + 2, r[1], r[2] + 2, vXi) + 2 * Nd] - nn[sub2ind(r[0] - 2, r[1], r[2] + 2, vXi) + 2 * Nd] - nn[sub2ind(r[0] + 2, r[1], r[2] - 2, vXi) + 2 * Nd] +
			(nn[sub2ind(r[0] + 1, r[1], r[2] - 2, vXi) + 2 * Nd] + nn[sub2ind(r[0] - 2, r[1], r[2] + 1, vXi) + 2 * Nd] - nn[sub2ind(r[0] - 1, r[1], r[2] - 2, vXi) + 2 * Nd] - nn[sub2ind(r[0] - 2, r[1], r[2] - 1, vXi) + 2 * Nd] +
				nn[sub2ind(r[0] + 2, r[1], r[2] - 1, vXi) + 2 * Nd] + nn[sub2ind(r[0] - 1, r[1], r[2] + 2, vXi) + 2 * Nd] - nn[sub2ind(r[0] + 1, r[1], r[2] + 2, vXi) + 2 * Nd] - nn[sub2ind(r[0] + 2, r[1], r[2] + 1, vXi) + 2 * Nd]) * 8.0f +
			(nn[sub2ind(r[0] + 1, r[1], r[2] + 1, vXi) + 2 * Nd] + nn[sub2ind(r[0] - 1, r[1], r[2] - 1, vXi) + 2 * Nd] - nn[sub2ind(r[0] + 1, r[1], r[2] - 1, vXi) + 2 * Nd] - nn[sub2ind(r[0] - 1, r[1], r[2] + 1, vXi) + 2 * Nd]) * 64.0f) / (144.0f * dr[0] * dr[2]);

		scalar nz011 = (nn[sub2ind(r[0], r[1] - 2, r[2] - 2, vXi) + 2 * Nd] + nn[sub2ind(r[0], r[1] + 2, r[2] + 2, vXi) + 2 * Nd] - nn[sub2ind(r[0], r[1] - 2, r[2] + 2, vXi) + 2 * Nd] - nn[sub2ind(r[0], r[1] + 2, r[2] - 2, vXi) + 2 * Nd] +
			(nn[sub2ind(r[0], r[1] + 1, r[2] - 2, vXi) + 2 * Nd] + nn[sub2ind(r[0], r[1] - 2, r[2] + 1, vXi) + 2 * Nd] - nn[sub2ind(r[0], r[1] - 1, r[2] - 2, vXi) + 2 * Nd] - nn[sub2ind(r[0], r[1] - 2, r[2] - 1, vXi) + 2 * Nd] +
				nn[sub2ind(r[0], r[1] + 2, r[2] - 1, vXi) + 2 * Nd] + nn[sub2ind(r[0], r[1] - 1, r[2] + 2, vXi) + 2 * Nd] - nn[sub2ind(r[0], r[1] + 1, r[2] + 2, vXi) + 2 * Nd] - nn[sub2ind(r[0], r[1] + 2, r[2] + 1, vXi) + 2 * Nd]) * 8.0f +
			(nn[sub2ind(r[0], r[1] + 1, r[2] + 1, vXi) + 2 * Nd] + nn[sub2ind(r[0], r[1] - 1, r[2] - 1, vXi) + 2 * Nd] - nn[sub2ind(r[0], r[1] + 1, r[2] - 1, vXi) + 2 * Nd] - nn[sub2ind(r[0], r[1] - 1, r[2] + 1, vXi) + 2 * Nd]) * 64.0f) / (144.0f * dr[1] * dr[2]);



		v100 = (-c1 * vv[sub2ind(r[0] + 2, r[1], r[2], vXi)] + c2 * vv[sub2ind(r[0] + 1, r[1], r[2], vXi)] - c2 * vv[sub2ind(r[0] - 1, r[1], r[2], vXi)] + c1 * vv[sub2ind(r[0] - 2, r[1], r[2], vXi)]) / dr[0];
		v010 = (-c1 * vv[sub2ind(r[0], r[1] + 2, r[2], vXi)] + c2 * vv[sub2ind(r[0], r[1] + 1, r[2], vXi)] - c2 * vv[sub2ind(r[0], r[1] - 1, r[2], vXi)] + c1 * vv[sub2ind(r[0], r[1] - 2, r[2], vXi)]) / dr[1];
		v001 = (-c1 * vv[sub2ind(r[0], r[1], r[2] + 2, vXi)] + c2 * vv[sub2ind(r[0], r[1], r[2] + 1, vXi)] - c2 * vv[sub2ind(r[0], r[1], r[2] - 1, vXi)] + c1 * vv[sub2ind(r[0], r[1], r[2] - 2, vXi)]) / dr[2];

		__syncthreads();

		scalar K = (k11 + k22 + k33) / 3.0;
		scalar Xi = 8.854 * (epar - eper) / K;
		scalar q0 = 2 * PI * chirality;
		scalar c0 = 2.5;

		// Reduced elastic constants
		k11 /= K;
		k22 /= K;
		k33 /= K;

		scalar nx000_new = ((k11 * nAvg[0][0]) / dr2[0] + (k33 * nAvg[1][0]) / dr2[1] + (k22 * nAvg[2][0]) / dr2[2] + k11 * ny110 - k33 * ny110 + (-k22 + k33) * nD[2][1] * (nD[1][0] - 2 * nD[0][1]) * nz000 + ((k22 - k33) * nAvg[1][0] * nz000*nz000) / dr2[1] +
			((-k22 + k33) * nAvg[2][0] * nz000*nz000) / dr2[2] + (-k22 + k33) * ny110 * nz000*nz000 + (-k22 + k33) * nD[1][0] * ny000 * nD[2][2] + (k22 - k33) * ny000 * nD[0][1] * nD[2][2] + (-k22 + k33) * nD[2][0] * ny000 * nD[1][2] + 2 * (k22 - k33) * ny000 * nD[1][2] * nD[0][2] + k11 * nz101 - k22 * nz101 +
			(k22 - k33) * nz000*nz000 * nz101 + (-k22 + k33) * nz000 * (2 * nx011 * ny000 + nD[2][0] * (nD[1][1] + 2 * nD[2][2]) - 2 * nD[1][0] * nD[1][2] + 3 * nD[0][1] * nD[1][2] - (nD[1][1] + 3 * nD[2][2]) * nD[0][2] - ny000 * (ny101 + nz110)) + 2 * k22 * nD[2][1] * q0 - 2 * k22 * nD[1][2] * q0 + (nz000 * v001 + ny000 * v010) * v100 * Xi)
			/ (c0 * (k11 / dr2[0] + (k33 + k22 * nz000*nz000 - k33 * nz000*nz000) / dr2[1] + (k22 - k22 * nz000*nz000 + k33 * nz000*nz000) / dr2[2]));

		scalar ny000_new = (k11 * nx110 - k33 * nx110 + (k33 * nAvg[0][1]) / dr2[0] + (k11 * nAvg[1][1]) / dr2[1] + (k22 * nAvg[2][1]) / dr2[2] + (k22 - k33) * nD[2][0] * (nD[1][0] - nD[0][1]) * nz000 + (-k22 + k33) * nx110 * nz000*nz000 + ((k22 - k33) * nAvg[0][1] * nz000*nz000) / dr2[0] +
			((-k22 + k33) * nAvg[2][1] * nz000*nz000) / dr2[2] + (-k22 + k33) * nx000 * nD[0][1] * nD[2][2] + (-k22 + k33) * nD[2][1] * nz000 * (nD[0][0] + nD[1][1] + 2 * nD[2][2]) + (k22 - k33) * nz000 * (nD[0][0] + nD[2][2]) * nD[1][2] + k11 * nz011 - k22 * nz011 + (-k22 + k33) * nx000 * nD[2][1] * nD[0][2] +
			(-k22 + k33) * (3 * nD[1][0] - 2 * nD[0][1]) * nz000 * nD[0][2] + 2 * (k22 - k33) * nx000 * nD[1][2] * nD[0][2] + (-k22 + k33) * nx000 * nz000 * (2 * ny101 - nz110) - 2 * k22 * nD[2][0] * q0 + 2 * k22 * nD[0][2] * q0 + v010 * (nz000 * v001 + nx000 * v100) * Xi) /
			((c0 * k33) / dr2[0] + (c0 * (dr2[2] * k11 + dr2[1] * k22 + (-dr[1] + dr[2]) * (dr[1] + dr[2]) * (k22 - k33) * nz000*nz000)) / (dr2[1] * dr2[2]) + ((k22 - k33) * nz000 * nAvg[0][2]) / dr2[0] +
				((k22 - k33) * (dr2[1] * (-(nx101 * nz000) + 2 * ny011 * nz000 + nD[1][1] * nD[2][2] + (nD[2][1] - 2 * nD[1][2]) * nD[1][2] - nD[2][0] * nD[0][2] + 2 * nD[0][2]*nD[0][2]) - nz000 * nAvg[1][2])) / dr2[1] + (-v010*v010 + v100*v100) * Xi);

		scalar nz000_new = (k33 * (-(nD[2][0] * nD[1][0] * ny000) + nx101 * (-1 + nx000*nx000 + ny000*ny000) + nx000 * nD[0][1] * (-nD[2][1] + nD[1][2]) + nx000 * (nD[1][1] + nD[2][2]) * nD[0][2] +
			ny000 * (-(nD[2][1] * nD[1][1]) + nD[2][0] * nD[0][1] + (nD[0][0] + 2 * nD[1][1] + nD[2][2]) * nD[1][2] + nD[1][0] * nD[0][2] - 2 * nD[0][1] * nD[0][2] + 2 * nx000 * nz110)) + ((k33 + k22 * ny000*ny000 - k33 * ny000*ny000) * nAvg[0][2]) / dr2[0] -
			((k33 + k22 * (-2 + nx000*nx000 + 2 * ny000*ny000) - k33 * (nx000*nx000 + 2 * ny000*ny000)) * nAvg[1][2]) / dr2[1] + ((k22 - k33) * (-1 + nx000*nx000 + ny000*ny000) * nAvg[2][2]) / dr2[2] + k11 * (nx101 + ny011 + nAvg[2][2] / dr2[2]) -
			k22 * (nx000*nx000 * nx101 + ny011 + nD[2][0] * ny000 * (-nD[1][0] + nD[0][1]) + ny000 * (nx101 * ny000 - nD[2][1] * nD[1][1] + (nD[0][0] + 2 * nD[1][1] + nD[2][2]) * nD[1][2] + (nD[1][0] - 2 * nD[0][1]) * nD[0][2]) + nx000 * (-(nD[2][1] * nD[0][1]) + nD[0][1] * nD[1][2] + (nD[1][1] + nD[2][2]) * nD[0][2] + 2 * ny000 * nz110) +
				2 * (-nD[1][0] + nD[0][1]) * q0) + v001 * (ny000 * v010 + nx000 * v100) * Xi) /
			(c0 * (k33 / dr2[0] + (k11 + (k22 - k33) * (-1 + nx000*nx000 + ny000*ny000)) / dr2[2] + (-(k22 * (-2 + nx000*nx000 + ny000*ny000)) + k33 * (-1 + nx000*nx000 + ny000*ny000)) / dr2[1]) + ((k22 - k33) * ny000 * nAvg[0][1]) / dr2[0] +
				((-k22 + k33) * ny000 * nAvg[1][1]) / dr2[1] - (k22 - k33) * (-nD[1][0]*nD[1][0] + nD[2][1]*nD[2][1] + nD[1][1]*nD[1][1] + 4 * nD[1][0] * nD[0][1] - 2 * nD[0][1]*nD[0][1] - nD[1][1] * nD[2][2] - nD[2][2] * (nD[0][0] + nD[2][2]) - nD[2][1] * nD[1][2] + nD[1][2]*nD[1][2] + ny000 * (nx110 - 2 * nz011) +
					nD[2][0] * (nD[2][0] - nD[0][2]) + nx000 * (ny110 - 2 * nz101)) + (-v001*v001 + v100*v100) * Xi);


		scalar nmag = nx000_new * nx000_new + ny000_new * ny000_new + nz000_new * nz000_new;

		nn[idx] = (1.0 + rate) * nx000_new / nmag - rate * nn[idx];
		nn[idx + Nd] = (1.0 + rate) * ny000_new / nmag - rate * nn[idx + Nd];
		nn[idx + Nd * 2] = (1.0 + rate) * nz000_new / nmag - rate * nn[idx + Nd * 2];
	}

	HEMI_DEV_CALLABLE
		void UpdateVoltageO4_Device(scalar* nn, scalar* vv, unsigned int idx, unsigned int Nd, const int* vXi, scalar epar, scalar eper, const scalar* dr, scalar rate) {
		using namespace LC::Cuda;

		int r[3];
		ind2sub(idx, vXi, r);

		for (int d = 0; d < 3; d++)
			if (r[d] < 2 || r[d] > vXi[d] - 3) return;

		constexpr scalar c1 = 1.0 / 12.0;
		constexpr scalar c2 = 2.0 / 3.0;

		scalar nx000 = nn[idx];
		scalar ny000 = nn[idx + Nd];
		scalar nz000 = nn[idx + 2 * Nd];
		scalar ea = epar - eper;

		scalar nx100 = (-c1 * nn[sub2ind(r[0] + 2, r[1], r[2], vXi)] + c2 * nn[sub2ind(r[0] + 1, r[1], r[2], vXi)] - c2 * nn[sub2ind(r[0] - 1, r[1], r[2], vXi)] + c1 * nn[sub2ind(r[0] - 2, r[1], r[2], vXi)]) / dr[0];
		scalar ny100 = (-c1 * nn[sub2ind(r[0] + 2, r[1], r[2], vXi) + Nd] + c2 * nn[sub2ind(r[0] + 1, r[1], r[2], vXi) + Nd] - c2 * nn[sub2ind(r[0] - 1, r[1], r[2], vXi) + Nd] + c1 * nn[sub2ind(r[0] - 2, r[1], r[2], vXi) + Nd]) / dr[0];
		scalar nz100 = (-c1 * nn[sub2ind(r[0] + 2, r[1], r[2], vXi) + 2 * Nd] + c2 * nn[sub2ind(r[0] + 1, r[1], r[2], vXi) + 2 * Nd] - c2 * nn[sub2ind(r[0] - 1, r[1], r[2], vXi) + 2 * Nd] + c1 * nn[sub2ind(r[0] - 2, r[1], r[2], vXi) + 2 * Nd]) / dr[0];

		scalar nx010 = (-c1 * nn[sub2ind(r[0], r[1] + 2, r[2], vXi)] + c2 * nn[sub2ind(r[0], r[1] + 1, r[2], vXi)] - c2 * nn[sub2ind(r[0], r[1] - 1, r[2], vXi)] + c1 * nn[sub2ind(r[0], r[1] - 2, r[2], vXi)]) / dr[1];
		scalar ny010 = (-c1 * nn[sub2ind(r[0], r[1] + 2, r[2], vXi) + Nd] + c2 * nn[sub2ind(r[0], r[1] + 1, r[2], vXi) + Nd] - c2 * nn[sub2ind(r[0], r[1] - 1, r[2], vXi) + Nd] + c1 * nn[sub2ind(r[0], r[1] - 2, r[2], vXi) + Nd]) / dr[1];
		scalar nz010 = (-c1 * nn[sub2ind(r[0], r[1] + 2, r[2], vXi) + 2 * Nd] + c2 * nn[sub2ind(r[0], r[1] + 1, r[2], vXi) + 2 * Nd] - c2 * nn[sub2ind(r[0], r[1] - 1, r[2], vXi) + 2 * Nd] + c1 * nn[sub2ind(r[0], r[1] - 2, r[2], vXi) + 2 * Nd]) / dr[1];

		scalar nx001 = (-c1 * nn[sub2ind(r[0], r[1], r[2] + 2, vXi)] + c2 * nn[sub2ind(r[0], r[1], r[2] + 1, vXi)] - c2 * nn[sub2ind(r[0], r[1], r[2] - 1, vXi)] + c1 * nn[sub2ind(r[0], r[1], r[2] - 2, vXi)]) / dr[2];
		scalar ny001 = (-c1 * nn[sub2ind(r[0], r[1], r[2] + 2, vXi) + Nd] + c2 * nn[sub2ind(r[0], r[1], r[2] + 1, vXi) + Nd] - c2 * nn[sub2ind(r[0], r[1], r[2] - 1, vXi) + Nd] + c1 * nn[sub2ind(r[0], r[1], r[2] - 2, vXi) + Nd]) / dr[2];
		scalar nz001 = (-c1 * nn[sub2ind(r[0], r[1], r[2] + 2, vXi) + 2 * Nd] + c2 * nn[sub2ind(r[0], r[1], r[2] + 1, vXi) + 2 * Nd] - c2 * nn[sub2ind(r[0], r[1], r[2] - 1, vXi) + 2 * Nd] + c1 * nn[sub2ind(r[0], r[1], r[2] - 2, vXi) + 2 * Nd]) / dr[2];


		scalar v100 = (-c1 * vv[sub2ind(r[0] + 2, r[1], r[2], vXi)] + c2 * vv[sub2ind(r[0] + 1, r[1], r[2], vXi)] - c2 * vv[sub2ind(r[0] - 1, r[1], r[2], vXi)] + c1 * vv[sub2ind(r[0] - 2, r[1], r[2], vXi)]) / dr[0];
		scalar v010 = (-c1 * vv[sub2ind(r[0], r[1] + 2, r[2], vXi)] + c2 * vv[sub2ind(r[0], r[1] + 1, r[2], vXi)] - c2 * vv[sub2ind(r[0], r[1] - 1, r[2], vXi)] + c1 * vv[sub2ind(r[0], r[1] - 2, r[2], vXi)]) / dr[1];
		scalar v001 = (-c1 * vv[sub2ind(r[0], r[1], r[2] + 2, vXi)] + c2 * vv[sub2ind(r[0], r[1], r[2] + 1, vXi)] - c2 * vv[sub2ind(r[0], r[1], r[2] - 1, vXi)] + c1 * vv[sub2ind(r[0], r[1], r[2] - 2, vXi)]) / dr[2];

		
		scalar v110 = (vv[sub2ind(r[0] - 2, r[1] - 2, r[2], vXi)] + vv[sub2ind(r[0] + 2, r[1] + 2, r[2], vXi)] - vv[sub2ind(r[0] - 2, r[1] + 2, r[2], vXi)] - vv[sub2ind(r[0] + 2, r[1] - 2, r[2], vXi)] +
			(vv[sub2ind(r[0] + 1, r[1] - 2, r[2], vXi)] + vv[sub2ind(r[0] - 2, r[1] + 1, r[2], vXi)] - vv[sub2ind(r[0] - 1, r[1] - 2, r[2], vXi)] - vv[sub2ind(r[0] - 2, r[1] - 1, r[2], vXi)] +
				vv[sub2ind(r[0] + 2, r[1] - 1, r[2], vXi)] + vv[sub2ind(r[0] - 1, r[1] + 2, r[2], vXi)] - vv[sub2ind(r[0] + 1, r[1] + 2, r[2], vXi)] - vv[sub2ind(r[0] + 2, r[1] + 1, r[2], vXi)]) * 8.0f +
			(vv[sub2ind(r[0] + 1, r[1] + 1, r[2], vXi)] + vv[sub2ind(r[0] - 1, r[1] - 1, r[2], vXi)] - vv[sub2ind(r[0] + 1, r[1] - 1, r[2], vXi)] - vv[sub2ind(r[0] - 1, r[1] + 1, r[2], vXi)]) * 64.0f) / (144.0f * dr[0] * dr[1]);


		scalar v101 = (vv[sub2ind(r[0] - 2, r[1], r[2] - 2, vXi)] + vv[sub2ind(r[0] + 2, r[1], r[2] + 2, vXi)] - vv[sub2ind(r[0] - 2, r[1], r[2] + 2, vXi)] - vv[sub2ind(r[0] + 2, r[1], r[2] - 2, vXi)] +
			(vv[sub2ind(r[0] + 1, r[1], r[2] - 2, vXi)] + vv[sub2ind(r[0] - 2, r[1], r[2] + 1, vXi)] - vv[sub2ind(r[0] - 1, r[1], r[2] - 2, vXi)] - vv[sub2ind(r[0] - 2, r[1], r[2] - 1, vXi)] +
				vv[sub2ind(r[0] + 2, r[1], r[2] - 1, vXi)] + vv[sub2ind(r[0] - 1, r[1], r[2] + 2, vXi)] - vv[sub2ind(r[0] + 1, r[1], r[2] + 2, vXi)] - vv[sub2ind(r[0] + 2, r[1], r[2] + 1, vXi)]) * 8.0f +
			(vv[sub2ind(r[0] + 1, r[1], r[2] + 1, vXi)] + vv[sub2ind(r[0] - 1, r[1], r[2] - 1, vXi)] - vv[sub2ind(r[0] + 1, r[1], r[2] - 1, vXi)] - vv[sub2ind(r[0] - 1, r[1], r[2] + 1, vXi)]) * 64.0f) / (144.0f * dr[0] * dr[2]);

		scalar v011 = (vv[sub2ind(r[0], r[1] - 2, r[2] - 2, vXi)] + vv[sub2ind(r[0], r[1] + 2, r[2] + 2, vXi)] - vv[sub2ind(r[0], r[1] - 2, r[2] + 2, vXi)] - vv[sub2ind(r[0], r[1] + 2, r[2] - 2, vXi)] +
			(vv[sub2ind(r[0], r[1] + 1, r[2] - 2, vXi)] + vv[sub2ind(r[0], r[1] - 2, r[2] + 1, vXi)] - vv[sub2ind(r[0], r[1] - 1, r[2] - 2, vXi)] - vv[sub2ind(r[0], r[1] - 2, r[2] - 1, vXi)] +
				vv[sub2ind(r[0], r[1] + 2, r[2] - 1, vXi)] + vv[sub2ind(r[0], r[1] - 1, r[2] + 2, vXi)] - vv[sub2ind(r[0], r[1] + 1, r[2] + 2, vXi)] - vv[sub2ind(r[0], r[1] + 2, r[2] + 1, vXi)]) * 8.0f +
			(vv[sub2ind(r[0], r[1] + 1, r[2] + 1, vXi)] + vv[sub2ind(r[0], r[1] - 1, r[2] - 1, vXi)] - vv[sub2ind(r[0], r[1] + 1, r[2] - 1, vXi)] - vv[sub2ind(r[0], r[1] - 1, r[2] + 1, vXi)]) * 64.0f) / (144.0f * dr[1] * dr[2]);
		

		scalar w200 = -2.5 / (dr[0] * dr[0]);
		scalar vm200 = (-c1 * vv[sub2ind(r[0] + 2, r[1], r[2], vXi)] + 2.0 * c2 * vv[sub2ind(r[0] + 1, r[1], r[2], vXi)] + 2.0 * c2 * vv[sub2ind(r[0] - 1, r[1], r[2], vXi)] - c1 * vv[sub2ind(r[0] - 2, r[1], r[2], vXi)]) / (dr[0] * dr[0]);

		scalar w020 = -2.5 / (dr[1] * dr[1]);
		scalar vm020 = (-c1 * vv[sub2ind(r[0], r[1] + 2, r[2], vXi)] + 2.0 * c2 * vv[sub2ind(r[0], r[1] + 1, r[2], vXi)] + 2.0 * c2 * vv[sub2ind(r[0], r[1] - 1, r[2], vXi)] - c1 * vv[sub2ind(r[0], r[1] - 2, r[2], vXi)]) / (dr[1] * dr[1]);

		scalar w002 = -2.5 / (dr[2] * dr[2]);
		scalar vm002 = (-c1 * vv[sub2ind(r[0], r[1], r[2] + 2, vXi)] + 2.0 * c2 * vv[sub2ind(r[0], r[1], r[2] + 1, vXi)] + 2.0 * c2 * vv[sub2ind(r[0], r[1], r[2] - 1, vXi)] - c1 * vv[sub2ind(r[0], r[1], r[2] - 2, vXi)]) / (dr[2] * dr[2]);

		__syncthreads();

		scalar vp = (-9. * ea * (-3. * (vm002 + vm020 + vm200) + nx100 * (ny000 * v010 + nz000 * v001 + 2. * nx000 * v100) +
			(nx000 * nx000) * vm200 + (ny000 * ny000) * vm020 + (nz000 * nz000) * vm002 + nx000 * ny010 * v100 + nx000 * ny100 * v010 +
			nx000 * nz001 * v100 + nx000 * nz100 * v001 + nx001 * nz000 * v100 + nx010 * ny000 * v100 + ny000 * nz001 * v010 + ny000 * nz010 * v001 +
			ny001 * nz000 * v010 + ny010 * nz000 * v001 + 2. * nx000 * ny000 * v110 + 2. * nx000 * nz000 * v101 + 2. * ny000 * ny010 * v010 + 2. * ny000 * nz000 * v011 +
			2. * nz000 * nz001 * v001) - 2. * (epar + 2. * eper) * (vm002 + vm020 + vm200)) / (2. * (epar + 2. * eper) * (w002 + w020 + w200) +
				9. * ea * ((-3. + nx000 * nx000) * w200 + (-3. + ny000 * ny000) * w020 + (-3. + nz000 * nz000) * w002));

		vv[idx] = (1. + rate) * vp - rate * vv[idx];
	}

	HEMI_DEV_CALLABLE
		void FreeEnergyDensityO2_Device(scalar *en, const scalar *nn, const scalar *vv, unsigned int idx, unsigned int Nd, const int* vXi, scalar k11, scalar k22, scalar k33, scalar ea, const scalar* dr, const scalar* dr2, scalar chirality) {
		using namespace LC::Cuda;

		int r[3];
		ind2sub(idx, vXi, r);

		for (int d = 0; d < 3; d++)
			if (r[d] == 0 || r[d] == vXi[d] - 1) return;


		scalar nx000 = nn[idx];
		scalar ny000 = nn[idx + Nd];
		scalar nz000 = nn[idx + 2 * Nd];
		scalar Xi = 8.8541878 * ea / (k11 + k22 + k33) * 3.;

		scalar nx100 = (nn[sub2ind(r[0] + 1, r[1], r[2], vXi)] - nn[sub2ind(r[0] + 1, r[1], r[2], vXi)]) / (2.0 * dr[0]);
		scalar ny100 = (nn[sub2ind(r[0] + 1, r[1], r[2], vXi) + Nd] - nn[sub2ind(r[0] + 1, r[1], r[2], vXi) + Nd]) / (2.0 * dr[0]);
		scalar nz100 = (nn[sub2ind(r[0] + 1, r[1], r[2], vXi) + 2 * Nd] - nn[sub2ind(r[0] + 1, r[1], r[2], vXi) + 2 * Nd]) / (2.0 * dr[0]);

		scalar nx010 = (nn[sub2ind(r[0], r[1] + 1, r[2], vXi)] - nn[sub2ind(r[0], r[1] - 1, r[2], vXi)]) / (2.0 * dr[1]);
		scalar ny010 = (nn[sub2ind(r[0], r[1] + 1, r[2], vXi) + Nd] - nn[sub2ind(r[0], r[1] - 1, r[2], vXi) + Nd]) / (2.0 * dr[1]);
		scalar nz010 = (nn[sub2ind(r[0], r[1] + 1, r[2], vXi) + 2 * Nd] - nn[sub2ind(r[0], r[1] - 1, r[2], vXi) + 2 * Nd]) / (2.0 * dr[1]);

		scalar nx001 = (nn[sub2ind(r[0], r[1], r[2] + 1, vXi)] - nn[sub2ind(r[0], r[1], r[2] - 1, vXi)]) / (2.0 * dr[2]);
		scalar ny001 = (nn[sub2ind(r[0], r[1], r[2] + 1, vXi) + Nd] - nn[sub2ind(r[0], r[1], r[2] - 1, vXi) + Nd]) / (2.0 * dr[2]);
		scalar nz001 = (nn[sub2ind(r[0], r[1], r[2] + 1, vXi) + 2 * Nd] - nn[sub2ind(r[0], r[1], r[2] - 1, vXi) + 2 * Nd]) / (2.0 * dr[2]);

		scalar nx200 = (nn[sub2ind(r[0] + 1, r[1], r[2], vXi)] + nn[sub2ind(r[0] - 1, r[1], r[2], vXi)] - 2.0 * nx000) / dr2[0];
		scalar ny200 = (nn[sub2ind(r[0] + 1, r[1], r[2], vXi) + Nd] + nn[sub2ind(r[0] - 1, r[1], r[2], vXi) + Nd] - 2.0 * ny000) / dr2[0];
		scalar nz200 = (nn[sub2ind(r[0] + 1, r[1], r[2], vXi) + 2 * Nd] + nn[sub2ind(r[0] - 1, r[1], r[2], vXi) + 2 * Nd] - 2.0 * nz000) / dr2[0];

		scalar nx020 = (nn[sub2ind(r[0], r[1] + 1, r[2], vXi)] + nn[sub2ind(r[0], r[1] - 1, r[2], vXi)] - 2.0 * nx000) / dr2[1];
		scalar ny020 = (nn[sub2ind(r[0], r[1] + 1, r[2], vXi) + Nd] + nn[sub2ind(r[0], r[1] - 1, r[2], vXi) + Nd] - 2.0 * ny000) / dr2[1];
		scalar nz020 = (nn[sub2ind(r[0], r[1] + 1, r[2], vXi) + 2 * Nd] + nn[sub2ind(r[0], r[1] - 1, r[2], vXi) + 2 * Nd] - 2.0 * nz000) / dr2[1];

		scalar nx002 = (nn[sub2ind(r[0], r[1], r[2] + 1, vXi)] + nn[sub2ind(r[0], r[1], r[2] - 1, vXi)] - 2.0 * nx000) / dr2[2];
		scalar ny002 = (nn[sub2ind(r[0], r[1], r[2] + 1, vXi) + Nd] + nn[sub2ind(r[0], r[1], r[2] - 1, vXi) + Nd] - 2.0 * ny000) / dr2[2];
		scalar nz002 = (nn[sub2ind(r[0], r[1], r[2] + 1, vXi) + 2 * Nd] + nn[sub2ind(r[0], r[1], r[2] - 1, vXi) + 2 * Nd] - 2.0 * nz000) / dr2[2];

		scalar v100 = (vv[sub2ind(r[0] + 1, r[1], r[2], vXi)] - vv[sub2ind(r[0] - 1, r[1], r[2], vXi)]) / (2.0 * dr[0]);
		scalar v010 = (vv[sub2ind(r[0], r[1] + 1, r[2], vXi)] - vv[sub2ind(r[0], r[1] - 1, r[2], vXi)]) / (2.0 * dr[1]);
		scalar v001 = (vv[sub2ind(r[0], r[1], r[2] + 1, vXi)] - vv[sub2ind(r[0], r[1], r[2] - 1, vXi)]) / (2.0 * dr[2]);

		en[idx] = (k11 * pow(nx100 + ny010 + nz001, 2) + k33 * (pow(nx000 * (-nx010 + ny100) + nz000 * (ny001 - nz010), 2) + pow(ny000 * (ny001 - nz010) + nx000 * (nx001 - nz100), 2) + pow(ny000 * (nx010 - ny100) + nz000 * (nx001 - nz100), 2)) -
			2 * k22 * (nx000 * (nx002 + nx020 + nx200) + pow(nx010 - ny100, 2) + ny000 * (ny002 + ny020 + ny200) + pow(nx100 + ny010 + nz001, 2) + pow(ny001 - nz010, 2) + pow(nx001 - nz100, 2) + nz000 * (nz002 + nz020 + nz200)) +
			k22 * pow((-nx010 + ny100) * nz000 + nx000 * (-ny001 + nz010) + ny000 * (nx001 - nz100) + 2.*PI*chirality, 2) - pow(nz000 * v001 + ny000 * v010 + nx000 * v100, 2) * Xi) / 2.;
		
	}

	HEMI_DEV_CALLABLE
		void FreeEnergyFunctionalDerivativeO2_Device(scalar* en_func_der, const scalar* nn, const scalar* vv, unsigned int idx, unsigned int Nd, const int* vXi, scalar k11, scalar k22, scalar k33, scalar ea, const scalar* dr, const scalar* dr2, scalar chirality) {
		using namespace LC::Cuda;

		int r[3];
		ind2sub(idx, vXi, r);

		for (int d = 0; d < 3; d++)
			if (r[d] == 0 || r[d] == vXi[d] - 1) return;


		scalar nx000 = nn[idx];
		scalar ny000 = nn[idx + Nd];
		scalar nz000 = nn[idx + 2 * Nd];
		scalar Xi = 8.8541878 * ea / (k11 + k22 + k33) * 3.;

		scalar nx100 = (nn[sub2ind(r[0] + 1, r[1], r[2], vXi)] - nn[sub2ind(r[0] + 1, r[1], r[2], vXi)]) / (2.0 * dr[0]);
		scalar ny100 = (nn[sub2ind(r[0] + 1, r[1], r[2], vXi) + Nd] - nn[sub2ind(r[0] + 1, r[1], r[2], vXi) + Nd]) / (2.0 * dr[0]);
		scalar nz100 = (nn[sub2ind(r[0] + 1, r[1], r[2], vXi) + 2 * Nd] - nn[sub2ind(r[0] + 1, r[1], r[2], vXi) + 2 * Nd]) / (2.0 * dr[0]);

		scalar nx010 = (nn[sub2ind(r[0], r[1] + 1, r[2], vXi)] - nn[sub2ind(r[0], r[1] - 1, r[2], vXi)]) / (2.0 * dr[1]);
		scalar ny010 = (nn[sub2ind(r[0], r[1] + 1, r[2], vXi) + Nd] - nn[sub2ind(r[0], r[1] - 1, r[2], vXi) + Nd]) / (2.0 * dr[1]);
		scalar nz010 = (nn[sub2ind(r[0], r[1] + 1, r[2], vXi) + 2 * Nd] - nn[sub2ind(r[0], r[1] - 1, r[2], vXi) + 2 * Nd]) / (2.0 * dr[1]);

		scalar nx001 = (nn[sub2ind(r[0], r[1], r[2] + 1, vXi)] - nn[sub2ind(r[0], r[1], r[2] - 1, vXi)]) / (2.0 * dr[2]);
		scalar ny001 = (nn[sub2ind(r[0], r[1], r[2] + 1, vXi) + Nd] - nn[sub2ind(r[0], r[1], r[2] - 1, vXi) + Nd]) / (2.0 * dr[2]);
		scalar nz001 = (nn[sub2ind(r[0], r[1], r[2] + 1, vXi) + 2 * Nd] - nn[sub2ind(r[0], r[1], r[2] - 1, vXi) + 2 * Nd]) / (2.0 * dr[2]);

		scalar nx200 = (nn[sub2ind(r[0] + 1, r[1], r[2], vXi)] + nn[sub2ind(r[0] - 1, r[1], r[2], vXi)] - 2.0 * nx000) / dr2[0];
		scalar ny200 = (nn[sub2ind(r[0] + 1, r[1], r[2], vXi) + Nd] + nn[sub2ind(r[0] - 1, r[1], r[2], vXi) + Nd] - 2.0 * ny000) / dr2[0];
		scalar nz200 = (nn[sub2ind(r[0] + 1, r[1], r[2], vXi) + 2 * Nd] + nn[sub2ind(r[0] - 1, r[1], r[2], vXi) + 2 * Nd] - 2.0 * nz000) / dr2[0];

		scalar nx020 = (nn[sub2ind(r[0], r[1] + 1, r[2], vXi)] + nn[sub2ind(r[0], r[1] - 1, r[2], vXi)] - 2.0 * nx000) / dr2[1];
		scalar ny020 = (nn[sub2ind(r[0], r[1] + 1, r[2], vXi) + Nd] + nn[sub2ind(r[0], r[1] - 1, r[2], vXi) + Nd] - 2.0 * ny000) / dr2[1];
		scalar nz020 = (nn[sub2ind(r[0], r[1] + 1, r[2], vXi) + 2 * Nd] + nn[sub2ind(r[0], r[1] - 1, r[2], vXi) + 2 * Nd] - 2.0 * nz000) / dr2[1];

		scalar nx002 = (nn[sub2ind(r[0], r[1], r[2] + 1, vXi)] + nn[sub2ind(r[0], r[1], r[2] - 1, vXi)] - 2.0 * nx000) / dr2[2];
		scalar ny002 = (nn[sub2ind(r[0], r[1], r[2] + 1, vXi) + Nd] + nn[sub2ind(r[0], r[1], r[2] - 1, vXi) + Nd] - 2.0 * ny000) / dr2[2];
		scalar nz002 = (nn[sub2ind(r[0], r[1], r[2] + 1, vXi) + 2 * Nd] + nn[sub2ind(r[0], r[1], r[2] - 1, vXi) + 2 * Nd] - 2.0 * nz000) / dr2[2];

		//nn110 = (nn[(rn.x * dims.y + rn.y) * dims.z + r.z] - nn[(rn.x * dims.y + rp.y) * dims.z + r.z] - nn[(rp.x * dims.y + rn.y) * dims.z + r.z] + nn[(rp.x * dims.y + rp.y) * dims.z + r.z]) / (4.0f * dr.x * dr.y);
		//nn101 = (nn[(rn.x * dims.y + r.y) * dims.z + rn.z] - nn[(rn.x * dims.y + r.y) * dims.z + rp.z] - nn[(rp.x * dims.y + r.y) * dims.z + rn.z] + nn[(rp.x * dims.y + r.y) * dims.z + rp.z]) / (4.0f * dr.x * dr.z);
		//nn011 = (nn[(r.x * dims.y + rn.y) * dims.z + rn.z] - nn[(r.x * dims.y + rn.y) * dims.z + rp.z] - nn[(r.x * dims.y + rp.y) * dims.z + rn.z] + nn[(r.x * dims.y + rp.y) * dims.z + rp.z]) / (4.0f * dr.y * dr.z);

		scalar nx110 = (nn[sub2ind(r[0] + 1, r[1] + 1, r[2], vXi)] - nn[sub2ind(r[0] + 1, r[1] - 1, r[2], vXi)]
			- nn[sub2ind(r[0] - 1, r[1] + 1, r[2], vXi)] + nn[sub2ind(r[0] - 1, r[1] - 1, r[2], vXi)]) / (4.0 * dr[0] * dr[1]);
		scalar ny110 = (nn[sub2ind(r[0] + 1, r[1] + 1, r[2], vXi) + Nd] - nn[sub2ind(r[0] + 1, r[1] - 1, r[2], vXi) + Nd]
			- nn[sub2ind(r[0] - 1, r[1] + 1, r[2], vXi) + Nd] + nn[sub2ind(r[0] - 1, r[1] - 1, r[2], vXi) + Nd]) / (4.0 * dr[0] * dr[1]);
		scalar nz110 = (nn[sub2ind(r[0] + 1, r[1] + 1, r[2], vXi) + 2 * Nd] - nn[sub2ind(r[0] + 1, r[1] - 1, r[2], vXi) + 2 * Nd]
			- nn[sub2ind(r[0] - 1, r[1] + 1, r[2], vXi) + 2 * Nd] + nn[sub2ind(r[0] - 1, r[1] - 1, r[2], vXi) + 2 * Nd]) / (4.0 * dr[0] * dr[1]);

		scalar nx101 = (nn[sub2ind(r[0]+1, r[1], r[2] + 1, vXi)] - nn[sub2ind(r[0] + 1, r[1], r[2] - 1, vXi)]
			- nn[sub2ind(r[0] - 1, r[1], r[2] + 1, vXi)] + nn[sub2ind(r[0] - 1, r[1], r[2] - 1, vXi)]) / (4.0 * dr[0] * dr[2]);
		scalar ny101 = (nn[sub2ind(r[0] + 1, r[1], r[2] + 1, vXi) + Nd] - nn[sub2ind(r[0] + 1, r[1], r[2] - 1, vXi) + Nd]
			- nn[sub2ind(r[0] - 1, r[1], r[2] + 1, vXi) + Nd] + nn[sub2ind(r[0] - 1, r[1], r[2] - 1, vXi) + Nd]) / (4.0 * dr[0] * dr[2]);
		scalar nz101 = (nn[sub2ind(r[0] + 1, r[1], r[2] + 1, vXi) + 2 * Nd] - nn[sub2ind(r[0] + 1, r[1], r[2] - 1, vXi) + 2 * Nd]
			- nn[sub2ind(r[0] - 1, r[1], r[2] + 1, vXi) + 2 * Nd] + nn[sub2ind(r[0] - 1, r[1], r[2] - 1, vXi) + 2 * Nd]) / (4.0 * dr[0] * dr[2]);

		scalar nx011 = (nn[sub2ind(r[0], r[1] + 1, r[2] + 1, vXi)] - nn[sub2ind(r[0], r[1] + 1, r[2] - 1, vXi)]
			- nn[sub2ind(r[0], r[1] - 1, r[2] + 1, vXi)] + nn[sub2ind(r[0], r[1] - 1, r[2] - 1, vXi)]) / (4.0 * dr[1] * dr[2]);
		scalar ny011 = (nn[sub2ind(r[0], r[1] + 1, r[2] + 1, vXi) + Nd] - nn[sub2ind(r[0], r[1] + 1, r[2] - 1, vXi) + Nd]
			- nn[sub2ind(r[0], r[1] - 1, r[2] + 1, vXi) + Nd] + nn[sub2ind(r[0], r[1] - 1, r[2] - 1, vXi) + Nd]) / (4.0 * dr[1] * dr[2]);
		scalar nz011 = (nn[sub2ind(r[0], r[1] + 1, r[2] + 1, vXi) + 2 * Nd] - nn[sub2ind(r[0], r[1] + 1, r[2] - 1, vXi) + 2 * Nd]
			- nn[sub2ind(r[0], r[1] - 1, r[2] + 1, vXi) + 2 * Nd] + nn[sub2ind(r[0], r[1] - 1, r[2] - 1, vXi) + 2 * Nd]) / (4.0 * dr[1] * dr[2]);


		scalar v100 = (vv[sub2ind(r[0] + 1, r[1], r[2], vXi)] - vv[sub2ind(r[0] - 1, r[1], r[2], vXi)]) / (2.0 * dr[0]);
		scalar v010 = (vv[sub2ind(r[0], r[1] + 1, r[2], vXi)] - vv[sub2ind(r[0], r[1] - 1, r[2], vXi)]) / (2.0 * dr[1]);
		scalar v001 = (vv[sub2ind(r[0], r[1], r[2] + 1, vXi)] - vv[sub2ind(r[0], r[1], r[2] - 1, vXi)]) / (2.0 * dr[2]);

		scalar q0 = 2. * PI * chirality;

		// |fsx|
		en_func_der[idx] = abs(k22 * nx002 - k33 * nx002 + k22 * nx020 - k33 * nx020 + k22 * nx200 + (2. * k22 - k33) * nx000 * pow(ny001, 2.) - k33 * nx010 * ny000 * ny010 + 2. * k33 * ny000 * ny010 * ny100 + k33 * nx000 * pow(ny100, 2.) + k33 * ny110 - (k22 - k33) * (nx020 - ny110) * pow(nz000, 2.) +
			(k22 - k33) * nx010 * ny000 * nz001 + (-k22 + k33) * ny000 * ny100 * nz001 + (k22 - k33) * nx001 * ny000 * nz010 + (2. * k22 - k33) * nx000 * pow(nz010, 2.) + (k22 - k33) * nx000 * ny000 * (ny002 - nz011) + 2. * (-k22 + k33) * ny000 * nz010 * nz100 + k33 * nx000 * pow(nz100, 2.) -
			(k22 - k33) * pow(ny000, 2.) * (nx002 - nz101) + k33 * nz101 - k11 * (nx200 + ny110 + nz101) + 2. * k22 * nz010 * q0 + ny001 * ((k22 - k33) * (nx010 - 2. * ny100) * nz000 + 2. * (-2. * k22 + k33) * nx000 * nz010 + ny000 * ((-2. * k22 + k33) * nx001 + (3 * k22 - k33) * nz100) - 2. * k22 * q0) -
			ny000 * v010 * v100 * Xi - nx000 * pow(v100, 2.) * Xi + nz000 * (nx001 * ((k22 - k33) * ny010 - k33 * nz001) + (-2. * k22 + k33) * nx010 * nz010 + (3 * k22 - k33) * ny100 * nz010 - (k22 - k33) * nx000 * (ny011 - nz020) + (-k22 + k33) * ny010 * nz100 + 2. * k33 * nz001 * nz100 +
				(k22 - k33) * ny000 * (2. * nx011 - ny101 - nz110) - v001 * v100 * Xi));

		// |fsy|
		en_func_der[idx] += abs(k33 * nx110 + (2. * k22 - k33) * pow(nx001, 2.) * ny000 + k22 * ny020 + k22 * ny200 - k33 * ny200 + (k22 - k33) * nx100 * ny001 * nz000 + (k22 - k33) * nx000 * ny100 * nz001 + 2. * (k22 - k33) * ny001 * nz000 * nz001 + (-k22 + k33) * nx100 * nz000 * nz010 + 2. * (-k22 + k33) * nz000 * nz001 * nz010 +
			(k22 - k33) * pow(ny000, 2.) * (ny002 - nz011) + (k22 - k33) * pow(nz000, 2.) * (nx110 + ny002 - ny200 - nz011) + k22 * nz011 - k11 * (nx110 + ny020 + nz011) + (k22 - k33) * nx000 * ny001 * nz100 + 3 * (k22 - k33) * nx010 * nz000 * nz100 + 2. * (-k22 + k33) * ny100 * nz000 * nz100 +
			2. * (-k22 + k33) * nx000 * nz010 * nz100 - (k22 - k33) * nx000 * nz000 * (nx011 - 2. * ny101 + nz110) - 2. * k22 * nz100 * q0 + nx001 * (-((k22 - k33) * (2. * nx010 - ny100) * nz000) + 2. * (-2. * k22 + k33) * ny000 * nz100 + 2. * k22 * q0) - nz000 * v001 * v010 * Xi - nx000 * v010 * v100 * Xi +
			ny000 * (k33 * pow(nx010, 2.) + (2. * k22 - k33) * pow(ny001, 2.) - 2. * k33 * nx010 * ny100 + k33 * pow(ny100, 2.) + (k22 - k33) * ny010 * nz001 + (-3 * k22 + k33) * ny001 * nz010 + k33 * pow(nz010, 2.) + (2. * k22 - k33) * pow(nz100, 2.) + (k22 - k33) * nx000 * (nx002 - nz101) -
				(k22 - k33) * nz000 * (nx101 - nz200) - pow(v010, 2.) * Xi));

		// |fsz|
		en_func_der[idx] += abs(k33 * nx101 + k22 * ny011 - k11 * (nx101 + ny011) + 2. * (-k22 + k33) * nx000 * ny001 * ny100 + (-k11 + k22) * nz002 + (k22 - k33) * nx000 * ny100 * nz010 - (k22 - k33) * pow(nz000, 2.) * (ny011 - nz020) + (k22 - k33) * nx000 * ny010 * nz100 + k22 * nz200 - k33 * nz200 -
			(k22 - k33) * pow(ny000, 2.) * (-nx101 + ny011 - nz020 + nz200) - 2. * k22 * nx010 * q0 + 2. * k22 * ny100 * q0 - nx000 * v001 * v100 * Xi +
			nz000 * (k33 * pow(nx001, 2.) + (2. * k22 - k33) * pow(nx010, 2.) + k33 * pow(ny001, 2.) + 2. * (-2. * k22 + k33) * nx010 * ny100 + (2. * k22 - k33) * pow(ny100, 2.) + (k22 - k33) * nx000 * (nx020 - ny110) - (k22 - k33) * ny000 * (nx110 - ny200) + (k22 - k33) * nx100 * nz001 +
				(k22 - k33) * ny010 * nz001 + (-3 * k22 + k33) * ny001 * nz010 + (2. * k22 - k33) * pow(nz010, 2.) - (k22 + k33) * nx001 * nz100 + k33 * pow(nz100, 2.) - pow(v001, 2.) * Xi) +
			ny000 * (-((k22 - k33) * (2. * ny001 * ny010 + 2. * nx001 * (nx010 - ny100) - (nx100 + 2. * ny010) * nz010 - (nx010 - 2. * ny100) * nz100 + nx000 * (nx011 + ny101 - 2. * nz110))) - v001 * v010 * Xi));


	}

	void FreeEnergyDensityO2(scalar* en_density, const scalar* directors, const scalar* voltage, const int* vXi, scalar k11, scalar k22, scalar k33, scalar ea, const scalar* dr, const scalar* dr2, scalar chirality, unsigned int N) {
	
		hemi::parallel_for(0u, N, [=] HEMI_LAMBDA(unsigned int idx) {
			FreeEnergyDensityO2_Device(en_density, directors, voltage, idx, N, vXi, k11, k22, k33, ea, dr, dr2, chirality);
		});
	}

	void FreeEnergyFunctionalDerivativeO2(scalar* en_density, const scalar* directors, const scalar* voltage, const int* vXi, scalar k11, scalar k22, scalar k33, scalar ea, const scalar* dr, const scalar* dr2, scalar chirality, unsigned int N) {

		hemi::parallel_for(0u, N, [=] HEMI_LAMBDA(unsigned int idx) {
			FreeEnergyFunctionalDerivativeO2_Device(en_density, directors, voltage, idx, N, vXi, k11, k22, k33, ea, dr, dr2, chirality);
		});
	}

	void OneConstAlgebraicO2(scalar* directors, scalar *voltage, const int* vXi, scalar K, scalar epar, scalar eper, const bool* bc, const scalar* cXi, const scalar* dr, const scalar* dr2, scalar chirality, scalar rate, unsigned int N) {

		hemi::parallel_for(0u, N, [=] HEMI_LAMBDA(unsigned int idx) {
			HandleBoundaryConditionsOrder2_Device(directors, voltage, idx, vXi, bc, N);
			OneConstAlgebraicO2_Device(directors, voltage, idx, N, vXi, K, epar, eper, dr, dr2, rate, chirality);
			UpdateVoltageO2_Device(directors, voltage, idx, N, vXi, epar, eper, dr, rate);
			Normalize_Device(directors, idx, N);
		});
	}


	void OneConstAlgebraicO4(scalar* directors, scalar *voltage, const int* vXi, scalar K, scalar epar, scalar eper, const bool* bc, const scalar* cXi, const scalar* dr, const scalar* dr2, scalar chirality, scalar rate, unsigned int N) {

		hemi::parallel_for(0u, N, [=] HEMI_LAMBDA(unsigned int idx) {
			HandleBoundaryConditionsOrder4_Device(directors, voltage, idx, vXi, bc, N);
			OneConstAlgebraicO4_Device(directors, voltage, idx, N, vXi, K, epar, eper, dr, dr2, rate, chirality);
			UpdateVoltageO4_Device(directors, voltage, idx, N, vXi, epar, eper, dr, rate);
			Normalize_Device(directors, idx, N);
		});
	}

	void ThreeConstAlgebraicO4(scalar* directors, scalar* voltage, const int* vXi, scalar k11, scalar k22, scalar k33, scalar epar, scalar eper, const bool* bc, const scalar* cXi, const scalar* dr, const scalar* dr2, scalar chirality, scalar rate, unsigned int N) {

		hemi::parallel_for(0u, N, [=] HEMI_LAMBDA(unsigned int idx) {
			HandleBoundaryConditionsOrder4_Device(directors, voltage, idx, vXi, bc, N);
			ThreeConstAlgebraicO4_Device(directors, voltage, idx, N, vXi, k11, k22, k33, epar, eper, dr, dr2, rate, chirality);
			UpdateVoltageO4_Device(directors, voltage, idx, N, vXi, epar, eper, dr, rate);
			Normalize_Device(directors, idx, N);
		});
	}

	/*
		TODO: ThreeConstAlgebraicO2_Device
	*/
	void ThreeConstAlgebraicO2(scalar* directors, scalar* voltage, const int* vXi, scalar k11, scalar k22, scalar k33, scalar epar, scalar eper, const bool* bc, const scalar* cXi, const scalar* dr, const scalar* dr2, scalar chirality, scalar rate, unsigned int N) {

		hemi::parallel_for(0u, N, [=] HEMI_LAMBDA(unsigned int idx) {
			HandleBoundaryConditionsOrder2_Device(directors, voltage, idx, vXi, bc, N);
			//ThreeConstAlgebraicO4_Device(directors, voltage, idx, N, vXi, k11, k22, k33, epar, eper, dr, dr2, rate, chirality);
			UpdateVoltageO2_Device(directors, voltage, idx, N, vXi, epar, eper, dr, rate);
			Normalize_Device(directors, idx, N);
		});
	}

	void UpdateVoltageO4GPU(scalar* directors, scalar* voltage, const int* vXi, scalar epar, scalar eper, const bool* bc, const scalar* dr, scalar rate, unsigned int N) {

		hemi::parallel_for(0u, N, [=] HEMI_LAMBDA(unsigned int idx) {
			HandleBoundaryConditionsOrder4_Device(directors, voltage, idx, vXi, bc, N);
			UpdateVoltageO4_Device(directors, voltage, idx, N, vXi, epar, eper, dr, rate);

		});
	}

	void UpdateVoltageO2GPU(scalar* directors, scalar* voltage, const int* vXi, scalar epar, scalar eper, const bool* bc, const scalar* dr, scalar rate, unsigned int N) {

		hemi::parallel_for(0u, N, [=] HEMI_LAMBDA(unsigned int idx) {
			HandleBoundaryConditionsOrder2_Device(directors, voltage, idx, vXi, bc, N);
			UpdateVoltageO2_Device(directors, voltage, idx, N, vXi, epar, eper, dr, rate);

		});
	}

	/* routine
		0 - FullFunctionalO2
		1 - OneConstFunctionalO2
		2 - FullAlgebraicO2
		3 - OneConstAlgebraicO2
		4 - FullFunctionalO4
		5 - OneConstFunctionalO4
		6 - FullAlgebraicO4
		7 - OneConstAlgebraicO4
	*/
	void RelaxGPU(scalar* directors, scalar *voltage, const int* vXi, scalar k11, scalar k22, scalar k33, scalar epar, scalar eper, const bool* bc, const scalar* cXi, scalar chirality, scalar rate, unsigned int iterations, int routine) {
		unsigned int N = vXi[0] * vXi[1] * vXi[2];

		hemi::Array<scalar> dirs(N * 3);
		hemi::Array<scalar> volt(N);
		hemi::Array<scalar> cX(3);
		hemi::Array<int> vX(3);
		hemi::Array<bool> BC(3);

		scalar K = (k11 + k22 + k33) / 3.0;

		int notificationIterations = iterations / 10;

		dirs.copyFromHost(directors, N * 3);
		volt.copyFromHost(voltage, N);
		cX.copyFromHost(cXi, 3);
		vX.copyFromHost(vXi, 3);
		BC.copyFromHost(bc, 3);

		hemi::Array<scalar> dr(3), dr2(3);
		{
			scalar* h_dr = dr.writeOnlyHostPtr();
			scalar* h_dr2 = dr2.writeOnlyHostPtr();
			for (int d = 0; d < 3; d++) {
				h_dr[d] = cXi[d] / (scalar)(vXi[d] - 1);
				h_dr2[d] = h_dr[d] * h_dr[d];
			}
		}

		// Flipped algebraic bit
		if (routine & 0x02) {
			// Flipped one const bit
			if (routine & 0x01) {
				typedef void(*method_t)(scalar*, scalar *, const int*, scalar, scalar, scalar, const bool*, const scalar*, const scalar*, const scalar*, scalar, scalar, unsigned int);
				method_t method;
				// Flipped order4 bit
				if (routine & 0x04) method = OneConstAlgebraicO4;
				else method = OneConstAlgebraicO2;

				for (unsigned int i = 0; i < iterations; i++) {
					method(dirs.devicePtr(),
						volt.devicePtr(),
						vX.readOnlyDevicePtr(),
						K,
						epar,
						eper,
						BC.readOnlyDevicePtr(),
						cX.readOnlyDevicePtr(),
						dr.readOnlyDevicePtr(),
						dr2.readOnlyDevicePtr(),
						chirality, rate, N);

					if ((i + 1) % notificationIterations == 0) {
						hipDeviceSynchronize();
						printf("Iterations = %d\n", i + 1);
					}
					

				}
			}
			else { // Three constant bit
				
				typedef void(*method_t)(scalar*, scalar*, const int*, scalar, scalar, scalar, scalar, scalar, const bool*, const scalar*, const scalar*, const scalar*, scalar, scalar, unsigned int);
				method_t method;
				// Flipped order4 bit
				if (routine & 0x04) method = ThreeConstAlgebraicO4;
				else method = ThreeConstAlgebraicO2;

				for (unsigned int i = 0; i < iterations; i++) {

					method(dirs.devicePtr(),
						volt.devicePtr(),
						vX.readOnlyDevicePtr(),
						k11,
						k22,
						k33,
						epar,
						eper,
						BC.readOnlyDevicePtr(),
						cX.readOnlyDevicePtr(),
						dr.readOnlyDevicePtr(),
						dr2.readOnlyDevicePtr(),
						chirality, rate, N);

					if ((i + 1) % notificationIterations == 0) {
						hipDeviceSynchronize();
						printf("Iterations = %d\n", i + 1);
					}
				
				}
			}

		}
		else {
			return;
		}
		hipDeviceSynchronize();
		hipMemcpy(directors, dirs.readOnlyHostPtr(), 3 * sizeof(scalar) * N, hipMemcpyDeviceToHost);
		hipMemcpy(voltage, volt.readOnlyHostPtr(), sizeof(scalar) * N, hipMemcpyDeviceToHost);
	}

	void UpdateVoltageGPU(scalar* directors, scalar* voltage, const int* vXi, scalar epar, scalar eper, const bool* bc, const scalar* cXi, scalar rate, unsigned int iterations, int routine) {
		unsigned int N = vXi[0] * vXi[1] * vXi[2];

		hemi::Array<scalar> dirs(N * 3);
		hemi::Array<scalar> volt(N);
		hemi::Array<int> vX(3);
		hemi::Array<bool> BC(3);

		dirs.copyFromHost(directors, N * 3);
		volt.copyFromHost(voltage, N);
		vX.copyFromHost(vXi, 3);
		BC.copyFromHost(bc, 3);

		hemi::Array<scalar> dr(3);
		{
			scalar* h_dr = dr.writeOnlyHostPtr();
			for (int d = 0; d < 3; d++) {
				h_dr[d] = cXi[d] / (scalar)(vXi[d] - 1);
			}
		}

		
		typedef void(*method_t)(scalar*, scalar*, const int*, scalar, scalar, const bool*, const scalar*, scalar, unsigned int);
		method_t method;
		// Flipped order4 bit
		if (routine & 0x04) method = UpdateVoltageO4GPU;
		else method = UpdateVoltageO2GPU;

		for (unsigned int i = 0; i < iterations; i++)
			method(dirs.devicePtr(),
				volt.devicePtr(),
				vX.readOnlyDevicePtr(),
				epar,
				eper,
				BC.readOnlyDevicePtr(),
				dr.readOnlyDevicePtr(),
				rate,
				N);
		
		hemi::synchronize();
		hipMemcpy(voltage, volt.readOnlyHostPtr(), sizeof(scalar) * N, hipMemcpyDeviceToHost);
	}

	void ComputeEnergyDensity(scalar* en_density, scalar* directors, scalar* voltage, const int* vXi, scalar k11, scalar k22, scalar k33, scalar epar, scalar eper, const scalar* cXi, scalar chirality) {
		
		unsigned int N = vXi[0] * vXi[1] * vXi[2];
		hemi::Array<scalar> dirs(N * 3);
		hemi::Array<scalar> volt(N);
		hemi::Array<scalar> en(N);
		hemi::Array<scalar> cX(3);
		hemi::Array<int> vX(3);

		scalar ea = epar - eper;

		dirs.copyFromHost(directors, N * 3);
		volt.copyFromHost(voltage, N);
		cX.copyFromHost(cXi, 3);
		vX.copyFromHost(vXi, 3);

		hemi::Array<scalar> dr(3), dr2(3);
		{
			scalar* h_dr = dr.writeOnlyHostPtr();
			scalar* h_dr2 = dr2.writeOnlyHostPtr();
			for (int d = 0; d < 3; d++) {
				h_dr[d] = cXi[d] / (scalar)(vXi[d] - 1);
				h_dr2[d] = h_dr[d] * h_dr[d];
			}
		}

		FreeEnergyDensityO2(
			en.devicePtr(),
			dirs.readOnlyDevicePtr(),
			volt.readOnlyDevicePtr(),
			vX.readOnlyDevicePtr(),
			k11,
			k22,
			k33,
			ea,
			dr.readOnlyDevicePtr(),
			dr2.readOnlyDevicePtr(),
			chirality,
			N);

		hemi::synchronize();
		hipMemcpy(en_density, en.readOnlyHostPtr(), sizeof(scalar) * N, hipMemcpyDeviceToHost);
	}

	void ComputeEnergyFunctionalDerivativeAbsSum(scalar* en_density, scalar* directors, scalar* voltage, const int* vXi, scalar k11, scalar k22, scalar k33, scalar epar, scalar eper, const scalar* cXi, scalar chirality) {
		unsigned int N = vXi[0] * vXi[1] * vXi[2];
		hemi::Array<scalar> dirs(N * 3);
		hemi::Array<scalar> volt(N);
		hemi::Array<scalar> en(N);
		hemi::Array<scalar> cX(3);
		hemi::Array<int> vX(3);

		scalar ea = epar - eper;

		dirs.copyFromHost(directors, N * 3);
		volt.copyFromHost(voltage, N);
		cX.copyFromHost(cXi, 3);
		vX.copyFromHost(vXi, 3);

		hemi::Array<scalar> dr(3), dr2(3);
		{
			scalar* h_dr = dr.writeOnlyHostPtr();
			scalar* h_dr2 = dr2.writeOnlyHostPtr();
			for (int d = 0; d < 3; d++) {
				h_dr[d] = cXi[d] / (scalar)(vXi[d] - 1);
				h_dr2[d] = h_dr[d] * h_dr[d];
			}
		}

		FreeEnergyFunctionalDerivativeO2(
			en.devicePtr(),
			dirs.readOnlyDevicePtr(),
			volt.readOnlyDevicePtr(),
			vX.readOnlyDevicePtr(),
			k11,
			k22,
			k33,
			ea,
			dr.readOnlyDevicePtr(),
			dr2.readOnlyDevicePtr(),
			chirality,
			N);

		hemi::synchronize();
		hipMemcpy(en_density, en.readOnlyHostPtr(), sizeof(scalar) * N, hipMemcpyDeviceToHost);
	}


}}

}}