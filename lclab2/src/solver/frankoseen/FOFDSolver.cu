#include "hip/hip_runtime.h"
#include "CudaContext.cuh"
#include "base/scalar.h"


namespace LC { namespace FrankOseen { namespace ElasticOnly {

	typedef void(*vFunction_t)(void* data, unsigned int);

	HEMI_DEV_CALLABLE
	void Normalize_Device(scalar* nn, unsigned int idx, unsigned int N) {
		scalar nx = nn[idx];
		scalar ny = nn[idx + N];
		scalar nz = nn[idx + N * 2];
		scalar len = sqrt(nx * nx + ny * ny + nz * nz);
		nn[idx] /= len;
		nn[idx + N] /= len;
		nn[idx + N * 2] /= len;
	}

	HEMI_DEV_CALLABLE
	void HandleBoundaryConditionsOrder2_Device(scalar *nn, unsigned int idx, const int * vXi, const bool *bc, unsigned int N) {
		using namespace LC::Cuda;

		int r[3];
		ind2sub(idx, vXi, r);
		
		for (int d = 0; d < 3; d++) {
			if (bc[0] && r[0] == 0) nn[idx + N * d] = nn[sub2ind(vXi[0] - 4, r[1], r[2], vXi) + N * d];
			else if (bc[0] && r[0] == 1) nn[idx + N * d] = nn[sub2ind(vXi[0] - 3, r[1], r[2], vXi) + N * d];
			else if (bc[0] && r[0] == vXi[0] - 2) nn[idx + N * d] = nn[sub2ind(2, r[1], r[2], vXi) + N * d];
			else if (bc[0] && r[0] == vXi[0] - 1) nn[idx + N * d] = nn[sub2ind(3, r[1], r[2], vXi) + N * d];

			if (bc[1] && r[1] == 0) nn[idx + N * d] = nn[sub2ind(r[0], vXi[1] - 4, r[2], vXi) + N * d];
			else if (bc[1] && r[1] == 1) nn[idx + N * d] = nn[sub2ind(r[0], vXi[1] - 3, r[2], vXi) + N * d];
			else if (bc[1] && r[1] == vXi[1] - 2) nn[idx + N * d] = nn[sub2ind(r[0], 2, r[2], vXi) + N * d];
			else if (bc[1] && r[1] == vXi[1] - 1) nn[idx + N * d] = nn[sub2ind(r[0], 3, r[2], vXi) + N * d];

			if (bc[2] && r[2] == 0) nn[idx + N * d] = nn[sub2ind(r[0], r[1], vXi[2] - 4, vXi) + N * d];
			else if (bc[2] && r[2] == 1) nn[idx + N * d] = nn[sub2ind(r[0], r[1], vXi[2] - 3, vXi) + N * d];
			else if (bc[2] && r[2] == vXi[2] - 2) nn[idx + N * d] = nn[sub2ind(r[0], r[1], 2, vXi) + N * d];
			else if (bc[2] && r[2] == vXi[2] - 1) nn[idx + N * d] = nn[sub2ind(r[0], r[1], 3, vXi) + N * d];
		}
	}


	// Update bulk nodes
	HEMI_DEV_CALLABLE
	void OneConstAlgebraicO2_Device(scalar* nn, unsigned int idx, unsigned int Nd, const int* vXi, const scalar* dr, const scalar *dr2, scalar rate, scalar chirality) {
		using namespace LC::Cuda;


		int r[3];
		ind2sub(idx, vXi, r);

		for (int d = 0; d < 3; d++)
			if (r[d] == 0 || r[d] == vXi[d] - 1) return;

		scalar N, curl;

		scalar nD[3][3];
		// [position][direction][front/back]
		scalar dir[3][3][2];
		scalar vol = 1.0, denom = 0.0;

		for (int d = 0; d < 3; d++) {
			vol *= dr2[d];
			denom += dr2[d] * dr2[(d + 1) % 3];

			// Fill
			dir[0][d][0] = nn[sub2ind(r[0] - 1, r[1], r[2], vXi) + Nd * d];
			dir[0][d][1] = nn[sub2ind(r[0] + 1, r[1], r[2], vXi) + Nd * d];

			dir[1][d][0] = nn[sub2ind(r[0], r[1] - 1, r[2], vXi) + Nd * d];
			dir[1][d][1] = nn[sub2ind(r[0], r[1] + 1, r[2], vXi) + Nd * d];

			dir[2][d][0] = nn[sub2ind(r[0], r[1], r[2] - 1, vXi) + Nd * d];
			dir[2][d][1] = nn[sub2ind(r[0], r[1], r[2] + 1, vXi) + Nd * d];

			nD[0][d] = (dir[0][d][1] - dir[0][d][0]) / (2.0 * dr[d]);
			nD[1][d] = (dir[1][d][1] - dir[1][d][0]) / (2.0 * dr[d]);
			nD[2][d] = (dir[2][d][1] - dir[2][d][0]) / (2.0 * dr[d]);
		}

		__syncthreads();

		scalar c = (1 + rate) * 1.0 / (2.0 * denom);

		for (int d = 0; d < 3; d++) {

			N = 0.0;

			for (int dd = 0; dd < 3; dd++)
				N += (vol / dr2[dd]) * (dir[dd][d][1] + dir[dd][d][0]);

			const int a = (d + 1) % 3;
			const int b = (d + 2) % 3;
			curl = nD[a][b] - nD[b][a];

			nn[idx + Nd * d] = c * (N - 4.0 * PI * chirality * vol * curl) - rate * nn[idx + Nd * d];
		}
	}

	void OneConstAlgebraicO2(scalar* directors, const int* vXi, const bool* bc, const scalar* cXi, const scalar *dr, const scalar *dr2, scalar chirality, scalar rate, unsigned int N) {
		
		hemi::parallel_for(0u, N, [=] HEMI_LAMBDA(unsigned int idx) {
			OneConstAlgebraicO2_Device(directors, idx, N, vXi, dr, dr2, rate, chirality);
			HandleBoundaryConditionsOrder2_Device(directors, idx, vXi, bc, N);
			Normalize_Device(directors, idx, N);
		});
	}

	// Add relax flag types somehow...
	void RelaxGPU(scalar* directors, const int *vXi, const bool *bc, const scalar *cXi, scalar chirality, scalar rate, unsigned int iterations) {
		unsigned int N = vXi[0] * vXi[1] * vXi[2];

		hemi::Array<scalar> dirs(N * 3);
		hemi::Array<scalar> cX(3);
		hemi::Array<int> vX(3);
		hemi::Array<bool> BC(3);

		dirs.copyFromHost(directors, N * 3);
		cX.copyFromHost(cXi, 3);
		vX.copyFromHost(vXi, 3);
		BC.copyFromHost(bc, 3);

		hemi::Array<scalar> dr(3), dr2(3);
		{
			scalar* h_dr = dr.writeOnlyHostPtr();
			scalar* h_dr2 = dr2.writeOnlyHostPtr();
			for (int d = 0; d < 3; d++) {
				h_dr[d] = cXi[d] / (scalar)(vXi[d] - 1);
				h_dr2[d] = h_dr[d] * h_dr[d];
			}
		}

		for (unsigned int i = 0; i < iterations; i++)
			OneConstAlgebraicO2(dirs.devicePtr(),
				vX.readOnlyDevicePtr(),
				BC.readOnlyDevicePtr(),
				cX.readOnlyDevicePtr(),
				dr.readOnlyDevicePtr(),
				dr2.readOnlyDevicePtr(),
				chirality, rate, N);
		hemi::synchronize();
		hipMemcpy(directors, dirs.readOnlyHostPtr(), 3 * sizeof(scalar) * N, hipMemcpyDeviceToHost);
	}


}}}