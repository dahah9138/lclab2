#include "hip/hip_runtime.h"
#include "CudaContext.cuh"
#include "scalar.h"

namespace LC { namespace FrankOseen {
	
namespace ElasticOnly { namespace RBF {

	// Tuple of derivatives

	struct Director {
		scalar nx = 0.0, ny = 0.0, nz = 0.0;
	};

	struct OneConstDerivatives {
		Director Dx, Dy, Dz, Dlap;
	};

	HEMI_DEV_CALLABLE
	OneConstDerivatives ComputeDerivatives(std::size_t index, scalar *directors, const std::size_t *neighbors,
		const scalar *dx, const scalar *dy, const scalar *dz, const scalar *lap, std::size_t N, std::size_t Nactive, int k) {

		OneConstDerivatives derivatives;
		int mapoffset = k * index;

		scalar nx, ny, nz;

		std::size_t nbh;
		for (int i = 0; i < k; i++) {

			nbh = neighbors[i * Nactive + index];

			nx = directors[nbh];
			ny = directors[nbh + N];
			nz = directors[nbh + 2 * N];

			derivatives.Dx.nx += dx[mapoffset + i] * nx;
			derivatives.Dx.ny += dx[mapoffset + i] * ny;
			derivatives.Dx.nz += dx[mapoffset + i] * nz;

			derivatives.Dy.nx += dy[mapoffset + i] * nx;
			derivatives.Dy.ny += dy[mapoffset + i] * ny;
			derivatives.Dy.nz += dy[mapoffset + i] * nz;

			derivatives.Dz.nx += dz[mapoffset + i] * nx;
			derivatives.Dz.ny += dz[mapoffset + i] * ny;
			derivatives.Dz.nz += dz[mapoffset + i] * nz;

			derivatives.Dlap.nx += lap[mapoffset + i] * nx;
			derivatives.Dlap.ny += lap[mapoffset + i] * ny;
			derivatives.Dlap.nz += lap[mapoffset + i] * nz;
		}

		return derivatives;
	}

	HEMI_DEV_CALLABLE
	void UpdateDirectorsAlgebraic(std::size_t glob_idx, scalar *directors, scalar lap, std::size_t N, const OneConstDerivatives &d, scalar chir, scalar rate) {

		scalar nx000 = directors[glob_idx];
		scalar ny000 = directors[glob_idx + N];
		scalar nz000 = directors[glob_idx + 2 * N];

		directors[glob_idx] = (1.0 + rate) / lap * (4.0 * PI * chir * (d.Dy.nz - d.Dz.ny) - d.Dlap.nx + lap * nx000) - rate * nx000;
		directors[glob_idx + N] = (1.0 + rate) / lap * (4.0 * PI * chir * (d.Dz.nx - d.Dx.nz) - d.Dlap.ny + lap * ny000) - rate * ny000;
		directors[glob_idx + 2 * N] = (1.0 + rate) / lap * (4.0 * PI * chir * (d.Dx.ny - d.Dy.nx) - d.Dlap.nz + lap * nz000) - rate * nz000;
	}

	HEMI_DEV_CALLABLE
	void Normalize(scalar *nn, std::size_t idx, std::size_t N) {
		scalar nx = nn[idx];
		scalar ny = nn[idx + N];
		scalar nz = nn[idx + N * 2];
		scalar len = sqrt(nx * nx + ny * ny + nz * nz);
		nn[idx] /= len;
		nn[idx + N] /= len;
		nn[idx + N * 2] /= len;
	}

	void OneConstAlgebraic(scalar * directors, const std::size_t * active_nodes, const std::size_t * neighbors, const scalar * dx, const scalar * dy, const scalar * dz, const scalar * lap,
		std::size_t N, std::size_t Nactive, int k, scalar chirality, scalar rate) {
		hemi::parallel_for(0u, Nactive, [=] HEMI_LAMBDA(unsigned int idx) {
			
			OneConstDerivatives derivatives = ComputeDerivatives(idx, directors, neighbors, dx, dy, dz, lap, N, Nactive, k);
			UpdateDirectorsAlgebraic(active_nodes[idx], directors, lap[k * idx], N, derivatives, chirality, rate);
			Normalize(directors, active_nodes[idx], N);

		});

	}



	void RelaxGPUOneConst(scalar* directors, const std::size_t* active_nodes, const std::size_t* neighbors, const scalar* dx, const scalar* dy, const scalar* dz, const scalar* lap,
		std::size_t N, std::size_t Nactive, std::size_t k, scalar chirality, scalar rate, std::size_t iterations) {

		hemi::Array<scalar> A_directors(N * 3);
		hemi::Array<std::size_t> A_active_nodes(Nactive);
		hemi::Array<std::size_t> A_neighbors(Nactive * k);
		hemi::Array<scalar> A_dx(Nactive * k);
		hemi::Array<scalar> A_dy(Nactive * k);
		hemi::Array<scalar> A_dz(Nactive * k);
		hemi::Array<scalar> A_lap(Nactive * k);

		A_directors.copyFromHost(directors, N * 3);
		A_active_nodes.copyFromHost(active_nodes, Nactive);
		A_neighbors.copyFromHost(neighbors, Nactive * k);
		A_dx.copyFromHost(dx, Nactive * k);
		A_dy.copyFromHost(dy, Nactive * k);
		A_dz.copyFromHost(dz, Nactive * k);
		A_lap.copyFromHost(lap, Nactive * k);


		typedef void(*method_t)(scalar*, const std::size_t*, const std::size_t*,
			const scalar*, const scalar*, const scalar*, const scalar*,
			std::size_t, std::size_t, int, scalar, scalar);
		method_t method;

		method = OneConstAlgebraic;


		for (int i = 0; i < iterations; i++) {
			// Call relax function
			method(A_directors.devicePtr(), A_active_nodes.readOnlyDevicePtr(), A_neighbors.readOnlyDevicePtr(),
				A_dx.readOnlyDevicePtr(), A_dy.readOnlyDevicePtr(), A_dz.readOnlyDevicePtr(), A_lap.readOnlyDevicePtr(), N, Nactive, k, chirality, rate);
		}

		hemi::synchronize();

		// Copy data back
		hipMemcpy(directors, A_directors.readOnlyHostPtr(), 3 * sizeof(scalar) * N, hipMemcpyDeviceToHost);
	}
}}

namespace Electric { namespace RBF {

	struct Director {
		scalar nx = 0.0, ny = 0.0, nz = 0.0;
	};

	struct VoltageDerivatives {
		scalar v100 = 0.0, v010 = 0.0, v001 = 0.0,
			v200 = 0.0, v020 = 0.0, v002 = 0.0,
			v110 = 0.0, v011 = 0.0, v101 = 0.0;
	};

	// Derivatives needed to update directors
	struct Derivatives {
		Director Dx, Dy, Dz, Dxx, Dyy, Dzz, Dxy, Dyz, Dzx;
		VoltageDerivatives Vd;
	};


	HEMI_DEV_CALLABLE
		Derivatives ComputeDerivatives(std::size_t index, scalar* directors, scalar *voltage, const std::size_t* neighbors,
			const scalar* dx, const scalar* dy, const scalar* dz,
			const scalar* dxx, const scalar* dyy, const scalar* dzz,
			const scalar* dxy, const scalar* dyz, const scalar* dzx,
			std::size_t N, std::size_t Nactive, int k) {

		Derivatives derivatives;
		int mapoffset = k * index;

		scalar nx, ny, nz, v;

		std::size_t nbh;
		for (int i = 0; i < k; i++) {

			nbh = neighbors[i * Nactive + index];

			nx = directors[nbh];
			ny = directors[nbh + N];
			nz = directors[nbh + 2 * N];

			v = voltage[nbh];

			derivatives.Dx.nx += dx[mapoffset + i] * nx;
			derivatives.Dx.ny += dx[mapoffset + i] * ny;
			derivatives.Dx.nz += dx[mapoffset + i] * nz;

			derivatives.Dy.nx += dy[mapoffset + i] * nx;
			derivatives.Dy.ny += dy[mapoffset + i] * ny;
			derivatives.Dy.nz += dy[mapoffset + i] * nz;

			derivatives.Dz.nx += dz[mapoffset + i] * nx;
			derivatives.Dz.ny += dz[mapoffset + i] * ny;
			derivatives.Dz.nz += dz[mapoffset + i] * nz;

			derivatives.Dxx.nx += dxx[mapoffset + i] * nx;
			derivatives.Dxx.ny += dxx[mapoffset + i] * ny;
			derivatives.Dxx.nz += dxx[mapoffset + i] * nz;

			derivatives.Dyy.nx += dyy[mapoffset + i] * nx;
			derivatives.Dyy.ny += dyy[mapoffset + i] * ny;
			derivatives.Dyy.nz += dyy[mapoffset + i] * nz;

			derivatives.Dzz.nx += dzz[mapoffset + i] * nx;
			derivatives.Dzz.ny += dzz[mapoffset + i] * ny;
			derivatives.Dzz.nz += dzz[mapoffset + i] * nz;

			derivatives.Dxy.nx += dxy[mapoffset + i] * nx;
			derivatives.Dxy.ny += dxy[mapoffset + i] * ny;
			derivatives.Dxy.nz += dxy[mapoffset + i] * nz;

			derivatives.Dyz.nx += dyz[mapoffset + i] * nx;
			derivatives.Dyz.ny += dyz[mapoffset + i] * ny;
			derivatives.Dyz.nz += dyz[mapoffset + i] * nz;

			derivatives.Dzx.nx += dzx[mapoffset + i] * nx;
			derivatives.Dzx.ny += dzx[mapoffset + i] * ny;
			derivatives.Dzx.nz += dzx[mapoffset + i] * nz;

			derivatives.Vd.v100 += dx[mapoffset + i] * v;
			derivatives.Vd.v010 += dy[mapoffset + i] * v;
			derivatives.Vd.v001 += dz[mapoffset + i] * v;

			// First component not necessary for these terms
			if (i > 0) {
				derivatives.Vd.v200 += dxx[mapoffset + i] * v;
				derivatives.Vd.v020 += dyy[mapoffset + i] * v;
				derivatives.Vd.v002 += dzz[mapoffset + i] * v;

				derivatives.Vd.v110 += dxy[mapoffset + i] * v;
				derivatives.Vd.v011 += dyz[mapoffset + i] * v;
				derivatives.Vd.v101 += dzx[mapoffset + i] * v;
			}

		}

		return derivatives;
	}

	HEMI_DEV_CALLABLE
	void UpdateDirectorsAlgebraic(std::size_t glob_idx, scalar *directors, scalar lap, std::size_t N, const Derivatives&d, scalar Xi, scalar chir, scalar rate) {

		scalar nx000 = directors[glob_idx];
		scalar ny000 = directors[glob_idx + N];
		scalar nz000 = directors[glob_idx + 2 * N];

		directors[glob_idx] = (1.0 + rate) / lap * (Xi * d.Vd.v100 * (ny000 * d.Vd.v010 + nz000 * d.Vd.v001) - 4.0 * PI * chir * (d.Dz.ny - d.Dy.nz) - d.Dxx.nx - d.Dyy.nx - d.Dzz.nx + lap * nx000) - rate * nx000;
		directors[glob_idx + N] = (1.0 + rate) / lap * (Xi * d.Vd.v010 * (nx000 * d.Vd.v100 + nz000 * d.Vd.v001) - 4.0 * PI * chir * (d.Dx.nz - d.Dz.nx) - d.Dxx.ny - d.Dyy.ny - d.Dzz.ny + lap * ny000) - rate * ny000;
		directors[glob_idx + 2 * N] = (1.0 + rate) / lap * (Xi * d.Vd.v001 * (nx000 * d.Vd.v100 + ny000 * d.Vd.v010) - 4.0 * PI * chir * (d.Dy.nx - d.Dx.ny) - d.Dxx.nz - d.Dyy.nz - d.Dzz.nz + lap * nz000) - rate * nz000;
	}

	HEMI_DEV_CALLABLE
	void Normalize(scalar *nn, std::size_t idx, std::size_t N) {
		scalar nx = nn[idx];
		scalar ny = nn[idx + N];
		scalar nz = nn[idx + N * 2];
		scalar len = sqrt(nx * nx + ny * ny + nz * nz);
		nn[idx] /= len;
		nn[idx + N] /= len;
		nn[idx + N * 2] /= len;
	}

	HEMI_DEV_CALLABLE
	void UpdateVoltageAlgebraic(std::size_t glob_idx, scalar* directors, scalar* voltage, std::size_t N, const Derivatives& d, scalar w100, scalar w010, scalar w001,
		scalar w200, scalar w020, scalar w002, scalar w110, scalar w011, scalar w101, scalar rate, scalar ea, scalar eper, scalar epar) {

		scalar nx000 = directors[glob_idx];
		scalar ny000 = directors[glob_idx + N];
		scalar nz000 = directors[glob_idx + 2 * N];

		voltage[glob_idx] = (1. + rate) * (-9. * ea * (-3. * (d.Vd.v002 + d.Vd.v020 + d.Vd.v200) + d.Dx.nx * (ny000 * d.Vd.v010 + nz000 * d.Vd.v001 + 
			2. * nx000 * d.Vd.v100) + (nx000 * nx000) * d.Vd.v200 + (ny000 * ny000) * d.Vd.v020 + 
			(nz000 * nz000) * d.Vd.v002 + nx000 * d.Dy.ny * d.Vd.v100 + nx000 * d.Dx.ny * d.Vd.v010 + 
			nx000 * d.Dz.nz * d.Vd.v100 + nx000 * d.Dx.nz * d.Vd.v001 + d.Dz.nx * nz000 * d.Vd.v100 + 
			d.Dy.nx * ny000 * d.Vd.v100 + ny000 * d.Dz.nz * d.Vd.v010 + ny000 * d.Dy.nz * d.Vd.v001 + 
			d.Dz.ny * nz000 * d.Vd.v010 + d.Dy.ny * nz000 * d.Vd.v001 + 2. * nx000 * ny000 * d.Vd.v110 + 
			2. * nx000 * nz000 * d.Vd.v101 + 2. * ny000 * d.Dy.ny * d.Vd.v010 + 2. * ny000 * nz000 * d.Vd.v011 + 
			2. * nz000 * d.Dz.nz * d.Vd.v001) - 2. * (epar + 2. * eper) * (d.Vd.v002 + d.Vd.v020 + 
				d.Vd.v200)) / (2. * (epar + 2. * eper) * (w002 + w020 + w200) + 9. * ea * (-3. * (w002 +
					w020 + w200) + d.Dx.nx * (ny000 * w010 + nz000 * w001 + 2. * nx000 * w100) +
					(nx000 * nx000) * w200 + (ny000 * ny000) * w020 + (nz000 * nz000) * w002 +
					nx000 * d.Dy.ny * w100 + nx000 * d.Dx.ny * w010 + nx000 * d.Dz.nz * w100 +
					nx000 * d.Dx.nz * w001 + d.Dz.nx * nz000 * w100 + d.Dy.nx * ny000 * w100 +
					ny000 * d.Dz.nz * w010 + ny000 * d.Dy.nz * w001 + d.Dz.ny * nz000 * w010 +
					d.Dy.ny * nz000 * w001 + 2. * nx000 * ny000 * w110 + 2. * nx000 * nz000 * w101 +
					2. * ny000 * d.Dy.ny * w010 + 2. * ny000 * nz000 * w011 + 2. * nz000 * d.Dz.nz * w001)) - rate * voltage[glob_idx];

	}

	void OneConstAlgebraic(scalar * directors, scalar* voltage, const std::size_t * active_nodes, const std::size_t * neighbors,
		const scalar * dx, const scalar * dy, const scalar * dz,
		const scalar* dxx, const scalar* dyy, const scalar* dzz,
		const scalar* dxy, const scalar* dyz, const scalar* dzx,
		std::size_t N, std::size_t Nactive, int k, scalar chirality, scalar rate, scalar ea, scalar eper, scalar epar, scalar Xi) {


		hemi::parallel_for(0u, Nactive, [=] HEMI_LAMBDA(unsigned int idx) {
			
			// Update directors
			Derivatives derivatives = ComputeDerivatives(idx, directors, voltage, neighbors, dx, dy, dz,
				dxx, dyy, dzz, dxy, dyz, dzx, N, Nactive, k);

			UpdateDirectorsAlgebraic(active_nodes[idx], directors, dxx[k * idx] + dyy[k * idx] + dzz[k * idx], N, derivatives, Xi, chirality, rate);
			Normalize(directors, active_nodes[idx], N);

			// Subtract first component for v100, v010, v001
			// to solve algebraic equation

			scalar v = voltage[active_nodes[idx]];

			derivatives.Vd.v100 -= dx[k * idx] * v;
			derivatives.Vd.v010 -= dy[k * idx] * v;
			derivatives.Vd.v001 -= dz[k * idx] * v;

			// Update voltage
			UpdateVoltageAlgebraic(active_nodes[idx], directors, voltage, N, derivatives, dx[k * idx], dy[k * idx], dz[k * idx],
				dxx[k * idx], dyy[k * idx], dzz[k * idx], dxy[k * idx], dyz[k * idx], dzx[k * idx], rate, ea, eper, epar);

		});
	}

	void EquilibriumVoltage(scalar* directors, scalar* voltage, const std::size_t* active_nodes, const std::size_t* neighbors,
		const scalar* dx, const scalar* dy, const scalar* dz,
		const scalar* dxx, const scalar* dyy, const scalar* dzz,
		const scalar* dxy, const scalar* dyz, const scalar* dzx,
		std::size_t N, std::size_t Nactive, int k, scalar chirality, scalar rate, scalar ea, scalar eper, scalar epar, scalar Xi) {


		hemi::parallel_for(0u, Nactive, [=] HEMI_LAMBDA(unsigned int idx) {

			// Update directors
			Derivatives derivatives = ComputeDerivatives(idx, directors, voltage, neighbors, dx, dy, dz,
				dxx, dyy, dzz, dxy, dyz, dzx, N, Nactive, k);

			// Subtract first component for v100, v010, v001
			// to solve algebraic equation

			scalar v = voltage[active_nodes[idx]];

			derivatives.Vd.v100 -= dx[k * idx] * v;
			derivatives.Vd.v010 -= dy[k * idx] * v;
			derivatives.Vd.v001 -= dz[k * idx] * v;

			// Update voltage
			UpdateVoltageAlgebraic(active_nodes[idx], directors, voltage, N, derivatives, dx[k * idx], dy[k * idx], dz[k * idx],
				dxx[k * idx], dyy[k * idx], dzz[k * idx], dxy[k * idx], dyz[k * idx], dzx[k * idx], rate, ea, eper, epar);

		});
	}



	void RelaxGPUOneConst(scalar* directors, scalar *voltage, const std::size_t* active_nodes, const std::size_t* neighbors,
		const scalar* dx, const scalar* dy, const scalar* dz,
		const scalar* dxx, const scalar* dyy, const scalar* dzz,
		const scalar* dxy, const scalar* dyz, const scalar* dzx,
		std::size_t N, std::size_t Nactive, std::size_t k, scalar chirality, scalar rate, scalar ea, scalar eper, scalar epar, scalar Xi, std::size_t iterations) {

		hemi::Array<scalar> A_directors(N * 3);
		hemi::Array<scalar> A_voltage(N);
		hemi::Array<std::size_t> A_active_nodes(Nactive);
		hemi::Array<std::size_t> A_neighbors(Nactive * k);
		hemi::Array<scalar> A_dx(Nactive * k);
		hemi::Array<scalar> A_dy(Nactive * k);
		hemi::Array<scalar> A_dz(Nactive * k);
		hemi::Array<scalar> A_dxx(Nactive * k);
		hemi::Array<scalar> A_dyy(Nactive * k);
		hemi::Array<scalar> A_dzz(Nactive * k);
		hemi::Array<scalar> A_dxy(Nactive * k);
		hemi::Array<scalar> A_dyz(Nactive * k);
		hemi::Array<scalar> A_dzx(Nactive * k);


		A_directors.copyFromHost(directors, N * 3);
		A_voltage.copyFromHost(voltage, N);
		A_active_nodes.copyFromHost(active_nodes, Nactive);
		A_neighbors.copyFromHost(neighbors, Nactive * k);
		A_dx.copyFromHost(dx, Nactive * k);
		A_dy.copyFromHost(dy, Nactive * k);
		A_dz.copyFromHost(dz, Nactive * k);
		A_dxx.copyFromHost(dxx, Nactive * k);
		A_dyy.copyFromHost(dyy, Nactive * k);
		A_dzz.copyFromHost(dzz, Nactive * k);
		A_dxy.copyFromHost(dxy, Nactive * k);
		A_dyz.copyFromHost(dyz, Nactive * k);
		A_dzx.copyFromHost(dzx, Nactive * k);


		typedef void(*method_t)(scalar*, scalar*, const std::size_t*, const std::size_t*,
			const scalar*, const scalar*, const scalar*,
			const scalar*, const scalar*, const scalar*,
			const scalar*, const scalar*, const scalar*,
			std::size_t, std::size_t, int, scalar, scalar, scalar, scalar, scalar, scalar);
		method_t method;

		method = OneConstAlgebraic;

		for (int i = 0; i < iterations; i++) {
			// Call relax function
			method(A_directors.devicePtr(), A_voltage.devicePtr(), A_active_nodes.readOnlyDevicePtr(), A_neighbors.readOnlyDevicePtr(),
				A_dx.readOnlyDevicePtr(), A_dy.readOnlyDevicePtr(), A_dz.readOnlyDevicePtr(),
				A_dxx.readOnlyDevicePtr(), A_dyy.readOnlyDevicePtr(), A_dzz.readOnlyDevicePtr(),
				A_dxy.readOnlyDevicePtr(), A_dyz.readOnlyDevicePtr(), A_dzx.readOnlyDevicePtr(),
				N, Nactive, k, chirality, rate, ea, eper, epar, Xi);
		}

		hemi::synchronize();

		// Copy data back
		hipMemcpy(directors, A_directors.readOnlyHostPtr(), 3 * sizeof(scalar) * N, hipMemcpyDeviceToHost);
		hipMemcpy(voltage, A_voltage.readOnlyHostPtr(), sizeof(scalar) * N, hipMemcpyDeviceToHost);
	}


	void FindEquilibriumVoltage(scalar* directors, scalar* voltage, const std::size_t* active_nodes, const std::size_t* neighbors,
		const scalar* dx, const scalar* dy, const scalar* dz,
		const scalar* dxx, const scalar* dyy, const scalar* dzz,
		const scalar* dxy, const scalar* dyz, const scalar* dzx,
		std::size_t N, std::size_t Nactive, std::size_t k, scalar chirality, scalar rate, scalar ea, scalar eper, scalar epar, scalar Xi, std::size_t iterations) {

		hemi::Array<scalar> A_directors(N * 3);
		hemi::Array<scalar> A_voltage(N);
		hemi::Array<std::size_t> A_active_nodes(Nactive);
		hemi::Array<std::size_t> A_neighbors(Nactive * k);
		hemi::Array<scalar> A_dx(Nactive * k);
		hemi::Array<scalar> A_dy(Nactive * k);
		hemi::Array<scalar> A_dz(Nactive * k);
		hemi::Array<scalar> A_dxx(Nactive * k);
		hemi::Array<scalar> A_dyy(Nactive * k);
		hemi::Array<scalar> A_dzz(Nactive * k);
		hemi::Array<scalar> A_dxy(Nactive * k);
		hemi::Array<scalar> A_dyz(Nactive * k);
		hemi::Array<scalar> A_dzx(Nactive * k);


		A_directors.copyFromHost(directors, N * 3);
		A_voltage.copyFromHost(voltage, N);
		A_active_nodes.copyFromHost(active_nodes, Nactive);
		A_neighbors.copyFromHost(neighbors, Nactive * k);
		A_dx.copyFromHost(dx, Nactive * k);
		A_dy.copyFromHost(dy, Nactive * k);
		A_dz.copyFromHost(dz, Nactive * k);
		A_dxx.copyFromHost(dxx, Nactive * k);
		A_dyy.copyFromHost(dyy, Nactive * k);
		A_dzz.copyFromHost(dzz, Nactive * k);
		A_dxy.copyFromHost(dxy, Nactive * k);
		A_dyz.copyFromHost(dyz, Nactive * k);
		A_dzx.copyFromHost(dzx, Nactive * k);

		for (int i = 0; i < iterations; i++) {

			EquilibriumVoltage(A_directors.devicePtr(), A_voltage.devicePtr(), A_active_nodes.readOnlyDevicePtr(), A_neighbors.readOnlyDevicePtr(),
				A_dx.readOnlyDevicePtr(), A_dy.readOnlyDevicePtr(), A_dz.readOnlyDevicePtr(),
				A_dxx.readOnlyDevicePtr(), A_dyy.readOnlyDevicePtr(), A_dzz.readOnlyDevicePtr(),
				A_dxy.readOnlyDevicePtr(), A_dyz.readOnlyDevicePtr(), A_dzx.readOnlyDevicePtr(),
				N, Nactive, k, chirality, rate, ea, eper, epar, Xi);
		}

		hemi::synchronize();

		// Copy data back
		hipMemcpy(voltage, A_voltage.readOnlyHostPtr(), sizeof(scalar) * N, hipMemcpyDeviceToHost);
	}

}}

}}